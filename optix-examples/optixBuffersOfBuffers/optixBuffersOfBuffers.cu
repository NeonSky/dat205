#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>
#include "commonStructs.h"
#include "helpers.h"

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};


//
// Declaration of our Buffer of Buffers
//
rtBuffer<rtBufferId<uchar4, 2> > Kd_layers;

rtBuffer<BasicLight>                 lights;
rtDeclareVariable(float3,            ambient_light_color, , );
rtDeclareVariable(unsigned int,      radiance_ray_type, , );
rtDeclareVariable(unsigned int,      shadow_ray_type, , );
rtDeclareVariable(rtObject,          top_object, , );
rtDeclareVariable(rtObject,          top_shadower, , );
rtDeclareVariable(float,             scene_epsilon, , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal,   attribute shading_normal, ); 
rtDeclareVariable(float3, texcoord,         attribute texcoord, ); 

rtDeclareVariable(optix::Ray,          ray,        rtCurrentRay, );
rtDeclareVariable(float,               t_hit,      rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd,        rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow, rtPayload, );

RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = optix::make_float3(0.0f);
  rtTerminateRay();
}


RT_PROGRAM void closest_hit_radiance()
{
  float3 world_shading_normal = optix::normalize( 
      rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = optix::normalize( 
      rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 normal = optix::faceforward(
      world_shading_normal, -ray.direction, world_geometric_normal );

  float3 hit_point = ray.origin + t_hit * ray.direction;
  
  //
  // Calculate Kd - loop over all nested buffers, accumulating color
  //
  float3 Kd = make_float3( 1.0f ); 
  for( int i = 0; i < Kd_layers.size(); ++i )
  {
    // Grab a refernce to the nested buffer so we dont need to perform
    // the buffer lookup multiple times 
    rtBufferId<uchar4, 2>& layer = Kd_layers[i];

    optix::size_t2 size  = layer.size();
    uint2  idx  = make_uint2( min( texcoord.x*size.x, size.x-1.0f ),
                              min( texcoord.y*size.y, size.y-1.0f ) );
    uchar4 val  = layer[ idx ]; 
    float4 fval = make_float4( val.x / 256.0f,
                               val.y / 256.0f,
                               val.z / 256.0f,
                               val.w / 256.0f );
    Kd = make_float3( fval )*fval.w + Kd*(1.0f - fval.w );
  }

  // ambient contribution
  float3 result = Kd * ambient_light_color;

  // compute direct lighting
  unsigned int num_lights = lights.size();
  for(int i = 0; i < num_lights; ++i) {
    BasicLight light = lights[i];
    float Ldist = optix::length(light.pos - hit_point);
    float3 L = optix::normalize(light.pos - hit_point);
    float nDl = optix::dot( normal, L);

    // cast shadow ray
    float3 light_attenuation = make_float3(static_cast<float>( nDl > 0.0f ));
    if ( nDl > 0.0f && light.casts_shadow ) {
      PerRayData_shadow shadow_prd;
      shadow_prd.attenuation = make_float3(1.0f);
      optix::Ray shadow_ray = optix::make_Ray(
          hit_point, L, shadow_ray_type, scene_epsilon, Ldist );
      rtTrace(top_shadower, shadow_ray, shadow_prd);
      light_attenuation = shadow_prd.attenuation;
    }

    // If not completely shadowed, light the hit point
    if( fmaxf(light_attenuation) > 0.0f ) {
      float3 Lc = light.color * light_attenuation;
      result += Kd * nDl * Lc;
    }
  }

  // pass the color back up the tree
  prd.result = result;
}
