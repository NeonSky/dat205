#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

#include "helpers.h"

using namespace optix;

rtDeclareVariable(float4, sphere, , );
rtDeclareVariable(float3, rotation, , );

rtDeclareVariable(float3, matrix_row_0, , );
rtDeclareVariable(float3, matrix_row_1, , );
rtDeclareVariable(float3, matrix_row_2, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 

RT_PROGRAM void intersect(int primIdx)
{
  float3 center = make_float3(sphere);
  float3 O = ray.origin - center;
  float3 D = ray.direction;
  float radius = sphere.w;

  float b = dot(O, D);
  float c = dot(O, O)-radius*radius;
  float disc = b*b-c;
  if(disc > 0.0f){
    float sdisc = sqrtf(disc);
    float root1 = (-b - sdisc);
    bool check_second = true;
    if( rtPotentialIntersection( root1 ) ) {
      shading_normal = geometric_normal = (O + root1*D)/radius;

      float3 polar;
      polar.x = dot(matrix_row_0, geometric_normal);
      polar.y = dot(matrix_row_1, geometric_normal);
      polar.z = dot(matrix_row_2, geometric_normal);
      polar = optix::cart_to_pol(polar);

      texcoord = make_float3( polar.x*0.5f*M_1_PIf, (polar.y+M_PI_2f)*M_1_PIf, polar.z/radius );

      if(rtReportIntersection(0))
        check_second = false;
    } 
    if(check_second) {
      float root2 = (-b + sdisc);
      if( rtPotentialIntersection( root2 ) ) {
        shading_normal = geometric_normal = (O + root2*D)/radius;

        float3 polar;
        polar.x = dot(matrix_row_0, geometric_normal);
        polar.y = dot(matrix_row_1, geometric_normal);
        polar.z = dot(matrix_row_2, geometric_normal);
        polar = optix::cart_to_pol(polar);

        texcoord = make_float3( polar.x*0.5f*M_1_PIf, (polar.y+M_PI_2f)*M_1_PIf, polar.z/radius );

        rtReportIntersection(0);
      }
    }
  }
}

RT_PROGRAM void bounds (int, optix::Aabb* aabb)
{
  const float3 cen = make_float3( sphere );
  const float3 rad = make_float3( sphere.w );
  
  if( rad.x > 0.0f && !isinf(rad.x) ) {
    aabb->m_min = cen - rad;
    aabb->m_max = cen + rad;
  } else {
    aabb->invalidate();
  }
}

