#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix_world.h>
#include "random.h"
#include "commonStructs.h"

using namespace optix;

rtDeclareVariable(float,       scene_epsilon, , );
rtDeclareVariable(float,       occlusion_distance, , );
rtDeclareVariable(int,         sqrt_occlusion_samples, , );
rtDeclareVariable(rtObject,    top_object, , );

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 

rtDeclareVariable(optix::Ray,   ray,          rtCurrentRay, );
rtDeclareVariable(float,        t_hit,        rtIntersectionDistance, );
rtDeclareVariable(uint2,        launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2,        launch_dim,   rtLaunchDim, );
rtDeclareVariable(unsigned int, frame, , );

rtDeclareVariable(unsigned int, subframe_idx, rtSubframeIndex, );

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_occlusion
{
  float occlusion;
};

rtDeclareVariable(PerRayData_radiance,  prd_radiance,  rtPayload, );
rtDeclareVariable(PerRayData_occlusion, prd_occlusion, rtPayload, );

RT_PROGRAM void closest_hit_radiance()
{
  float3 phit = ray.origin + t_hit * ray.direction;

  float3 world_shading_normal   = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
  float3 world_geometric_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
  float3 ffnormal = faceforward(world_shading_normal, -ray.direction, world_geometric_normal);

  optix::Onb onb(ffnormal);

  unsigned int seed = tea<4>( launch_dim.x*launch_index.y+launch_index.x, frame + subframe_idx );

  float       result           = 0.0f;
  const float inv_sqrt_samples = 1.0f / float(sqrt_occlusion_samples);
  for( int i=0; i<sqrt_occlusion_samples; ++i ) {
    for( int j=0; j<sqrt_occlusion_samples; ++j ) {

      PerRayData_occlusion prd_occ;
      prd_occ.occlusion = 0.0f;

      // Stratify samples via simple jitterring
      float u1 = (float(i) + rnd( seed ) )*inv_sqrt_samples;
      float u2 = (float(j) + rnd( seed ) )*inv_sqrt_samples;

      float3 dir;
      optix::cosine_sample_hemisphere( u1, u2, dir );
      onb.inverse_transform( dir );

      optix::Ray occlusion_ray = optix::make_Ray( phit, dir, 1, scene_epsilon,
                                                  occlusion_distance );
      rtTrace( top_object, occlusion_ray, prd_occ );

      result += 1.0f-prd_occ.occlusion;
    }
  }

  result /= (float)(sqrt_occlusion_samples*sqrt_occlusion_samples);


  prd_radiance.result = make_float3(result);
}

RT_PROGRAM void any_hit_occlusion()
{
  prd_occlusion.occlusion = 1.0f;

  rtTerminateRay();
}



