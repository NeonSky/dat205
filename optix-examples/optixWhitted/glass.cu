#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"

using namespace optix;

rtDeclareVariable(rtObject,     top_object, , );
rtDeclareVariable(float,        scene_epsilon, , );
rtDeclareVariable(int,          max_depth, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(float,        importance_cutoff, , );
rtDeclareVariable(float3,       cutoff_color, , );
rtDeclareVariable(float,        fresnel_exponent, , );
rtDeclareVariable(float,        fresnel_minimum, , );
rtDeclareVariable(float,        fresnel_maximum, , );
rtDeclareVariable(float,        refraction_index, , );
rtDeclareVariable(int,          refraction_maxdepth, , );
rtDeclareVariable(int,          reflection_maxdepth, , );
rtDeclareVariable(float3,       refraction_color, , );
rtDeclareVariable(float3,       reflection_color, , );
rtDeclareVariable(float3,       extinction_constant, , );

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );

// -----------------------------------------------------------------------------

static __device__ __inline__ float3 TraceRay(float3 origin, float3 direction, int depth, float importance )
{
  optix::Ray ray = optix::make_Ray( origin, direction, radiance_ray_type, 0.0f, RT_DEFAULT_MAX );
  PerRayData_radiance prd;
  prd.depth = depth;
  prd.importance = importance;

  rtTrace( top_object, ray, prd );
  return prd.result;
}

static __device__ __inline__ float3 exp( const float3& x )
{
  return make_float3(exp(x.x), exp(x.y), exp(x.z));
}

// -----------------------------------------------------------------------------

RT_PROGRAM void closest_hit_radiance()
{
  // intersection vectors
  const float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal
  const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);
  const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);
  const float3 i = ray.direction;                                            // incident direction
        float3 t;                                                            // transmission direction
        float3 r;                                                            // reflection direction

  float reflection = 1.0f;
  float3 result = make_float3(0.0f);
  
  const int depth = prd_radiance.depth;

  float3 beer_attenuation;
  if(dot(n, ray.direction) > 0) {
    // Beer's law attenuation
    beer_attenuation = exp(extinction_constant * t_hit);
  } else {
    beer_attenuation = make_float3(1);
  }

  // refraction
  if (depth < min(refraction_maxdepth, max_depth))
  {
    if ( refract(t, i, n, refraction_index) )
    {
      // check for external or internal reflection
      float cos_theta = dot(i, n);
      if (cos_theta < 0.0f)
        cos_theta = -cos_theta;
      else
        cos_theta = dot(t, n);

      reflection = fresnel_schlick(cos_theta, fresnel_exponent, fresnel_minimum, fresnel_maximum);

      float importance = prd_radiance.importance * (1.0f-reflection) * optix::luminance( refraction_color * beer_attenuation );
      float3 color = cutoff_color;
      if ( importance > importance_cutoff ) {
        color = TraceRay(bhp, t, depth+1, importance);
      }
      result += (1.0f - reflection) * refraction_color * color;
    }
    // else TIR
  } // else reflection==1 so refraction has 0 weight

  // reflection
  float3 color = cutoff_color;
  if (depth < min(reflection_maxdepth, max_depth))
  {
    r = reflect(i, n);
  
    float importance = prd_radiance.importance * reflection * optix::luminance( reflection_color * beer_attenuation );
    if ( importance > importance_cutoff ) {
      color = TraceRay( fhp, r, depth+1, importance );
    }
  }
  result += reflection * reflection_color * color;

  result = result * beer_attenuation;

  prd_radiance.result = result;
}

// -----------------------------------------------------------------------------

//
// Attenuates shadow rays for shadowing transparent objects
//
rtDeclareVariable(float3, shadow_attenuation, , );

RT_PROGRAM void any_hit_shadow()
{
  float3 world_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float nDi = fabs(dot(world_normal, ray.direction));

  prd_shadow.attenuation *= 1-fresnel_schlick(nDi, 5, 1-shadow_attenuation, make_float3(1));
  if(optix::luminance(prd_shadow.attenuation) < importance_cutoff)
    rtTerminateRay();
  else
    rtIgnoreIntersection();
}
