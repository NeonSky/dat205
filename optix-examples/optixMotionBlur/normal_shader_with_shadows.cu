#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "commonStructs.h"

using namespace optix;

rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 

rtDeclareVariable(unsigned int,      radiance_ray_type, , );
rtDeclareVariable(unsigned int,      shadow_ray_type, , );
rtDeclareVariable(float,             scene_epsilon, , );
rtDeclareVariable(rtObject,          top_object, , );
rtDeclareVariable(rtObject,          top_shadower, , );
rtBuffer<BasicLight>                 lights;

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

struct PerRayData_shadow
{
  float3 attenuation;
};

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow,   prd_shadow,   rtPayload, );


RT_PROGRAM void any_hit_shadow()
{
  // this material is opaque, so it fully attenuates all shadow rays
  prd_shadow.attenuation = make_float3(0);

  rtTerminateRay();
}

RT_PROGRAM void closest_hit_radiance()
{
  
  const float3 hit_point = ray.origin + t_hit * ray.direction;

  const float3 world_shading_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  const float3 ffnormal = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  const float3 Kd = world_shading_normal*0.5f + make_float3(0.5f);

  float3 result = make_float3( 0.0f );
  unsigned int num_lights = lights.size();
  for(int i = 0; i < num_lights; ++i) {
      BasicLight light = lights[i];
      float3 L = optix::normalize(light.pos - hit_point);
      float Ldist = optix::length( L );
      float NdotL = optix::dot( ffnormal, L );

      // cast shadow ray
      if ( NdotL > 0.0f ) {
          float3 light_attenuation = make_float3( 1.0f );
          if ( light.casts_shadow ) {
              PerRayData_shadow shadow_prd;
              shadow_prd.attenuation = make_float3( 1.0f );
              optix::Ray shadow_ray = optix::make_Ray( hit_point, L, /*shadow_ray_type*/ 1, scene_epsilon, Ldist );
              rtTrace(top_shadower, shadow_ray, shadow_prd);
              light_attenuation = shadow_prd.attenuation;
          }
          result += Kd * light_attenuation * light.color;
      }
  }

  prd_radiance.result = result;
}

