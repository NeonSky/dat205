#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

rtBuffer<float4, 2> sysOutputBuffer; // RGBA32F

rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(float3, sysColorBackground, , );

// Entry point for simple color filling kernel.
RT_PROGRAM void raygeneration()
{
  sysOutputBuffer[theLaunchIndex] = make_float4(sysColorBackground, 1.0f);
}