#include "hip/hip_runtime.h"
/* 
* Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
*
* Redistribution and use in source and binary forms, with or without
* modification, are permitted provided that the following conditions
* are met:
*  * Redistributions of source code must retain the above copyright
*    notice, this list of conditions and the following disclaimer.
*  * Redistributions in binary form must reproduce the above copyright
*    notice, this list of conditions and the following disclaimer in the
*    documentation and/or other materials provided with the distribution.
*  * Neither the name of NVIDIA CORPORATION nor the names of its
*    contributors may be used to endorse or promote products derived
*    from this software without specific prior written permission.
*
* THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
* EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
* IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
* PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
* CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
* EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
* PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
* PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
* OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
* (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
* OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_aabb_namespace.h>

using namespace optix;

rtBuffer<float4>    positions_buffer;

rtDeclareVariable(float2,       particle_rbf,    attribute particle_rbf, );
rtDeclareVariable(optix::Ray,   ray,                rtCurrentRay, );
rtDeclareVariable(float,        fixed_radius, ,  );


RT_PROGRAM void particle_intersect( int primIdx )
{
    const float4 pos = positions_buffer[primIdx];
    const float3 pos3 = make_float3(pos.x, pos.y, pos.z);
    const float t = length(pos3 - ray.origin);
    const float3 samplePos = ray.origin + ray.direction * t;

    if( (length(pos3 - samplePos) < fixed_radius) && rtPotentialIntersection(t) )
    {
      particle_rbf.x = t;
      particle_rbf.y = __int_as_float(primIdx);
      rtReportIntersection( 0 );
    }
}

//for accel build
RT_PROGRAM void particle_bounds( int primIdx, float result[6] )
{
    const float4 position = positions_buffer[ primIdx ];
    const float radius = fixed_radius;

    optix::Aabb *aabb = (optix::Aabb *) result;

    aabb->m_min.x = position.x - radius;
    aabb->m_min.y = position.y - radius;
    aabb->m_min.z = position.z - radius;

    aabb->m_max.x = position.x + radius;
    aabb->m_max.y = position.y + radius;
    aabb->m_max.z = position.z + radius;
}

