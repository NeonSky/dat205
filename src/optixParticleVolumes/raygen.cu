#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "random.h"
#include "commonStructs.h"
#include "transferFunction.h"


using namespace optix;

rtBuffer<float4>    positions_buffer;

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<float4, 2>              accum_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );
rtDeclareVariable(unsigned int,  frame, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(int,           tf_type, ,  );
rtDeclareVariable(float,         fixed_radius, ,  );
rtDeclareVariable(float3,        bbox_min, , );
rtDeclareVariable(float3,        bbox_max, , );

rtDeclareVariable(float,         opacity, , );
rtDeclareVariable(float,         particlesPerSlab, , );
rtDeclareVariable(float,         wScale, , );
rtDeclareVariable(float,         redshift, , );


RT_PROGRAM void raygen_program()
{

  size_t2 screen = output_buffer.size();
  unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame);

  float2 subpixel_jitter = make_float2(0.0f, 0.0f);

  float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);

  const float redshiftScale = redshift / length(bbox_max - bbox_min);

  optix::Ray ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData prd;

  //ray-AABB intersection to determine number of segments
 
  float3 t0, t1, tmin, tmax;
  t0 = (bbox_max - ray_origin) / ray_direction;
  t1 = (bbox_min - ray_origin) / ray_direction;
  tmax = fmaxf(t0, t1);
  tmin = fminf(t0, t1);
  float tenter = fmaxf(0.f, fmaxf(tmin.x, fmaxf(tmin.y, tmin.z)));
  float texit = fminf(tmax.x, fminf(tmax.y, tmax.z));

  float slab_spacing = PARTICLE_BUFFER_SIZE * particlesPerSlab * fixed_radius;

  float3 result = make_float3(0);
  float result_alpha = 0.f;
  
  if (tenter < texit)
  {
    float tbuffer = 0.f;

    //for each segment, 
    //  traverse the BVH (collect deep samples in prd.particles), 
    //  sort,
    //  integrate.
    
    while(tbuffer < texit && result_alpha < 0.97f)
    {
      prd.tail = 0;
      ray.tmin = fmaxf(tenter, tbuffer);
      ray.tmax = fminf(texit, tbuffer + slab_spacing);

      if (ray.tmax > tenter)    //doing this will keep rays more coherent
      {
        rtTrace(top_object, ray, prd);

        //sort() in RT Gems pseudocode
        int N = prd.tail;
#if (PARTICLE_BUFFER_SIZE <= 64)
        //bubble sort
        for(int i=0; i<N; i++)
          for(int j=0; j < N-i-1; j++)
          {
            const float2 tmp = prd.particles[i];
            if( tmp.x < prd.particles[j].x) {
              prd.particles[i] = prd.particles[j];
              prd.particles[j] = tmp;
            }
          }
#else
        //bitonic sort
        int Nup2 = 1;
        while (Nup2 < N)
          Nup2 = Nup2 << 1;
        Nup2 = min(Nup2, PARTICLE_BUFFER_SIZE_SPLAT);

        //power of two clamp
        for(int i=N; i<Nup2; i++)
          prd.particles[i].x = 1e20f;
        N = Nup2;

        for (int k=2; k<=N; k=k<<1) {
          for (int j=k>>1; j>0; j=j>>1) {
            for (int i=0; i<N; i++) {
              const int ij=i^j;
              if (ij>i) {
                const int ik = i&k;
                const float2 tmp = prd.particles[i];
                if (ik==0 && tmp.x > prd.particles[ij].x) {   //sort ascending
                  prd.particles[i] = prd.particles[ij];
                  prd.particles[ij] = tmp;
                }
                if (ik!=0 && tmp.x < prd.particles[ij].x) {   //sort descending
                  prd.particles[i] = prd.particles[ij];
                  prd.particles[ij] = tmp;
                }
              }
            }
          }
        }
#endif
        const float inv_fixed_radius_scale = 2.f / fixed_radius;

        //integrate depth-sorted list of particles
        for(int i=0; i<prd.tail; i++) {

          float trbf = prd.particles[i].x;
          int idx = __float_as_int(prd.particles[i].y);
          float3 hit_sample = ray.origin + ray.direction * trbf;

          float4 pos = positions_buffer[idx];
          float3 hit_normal = make_float3(pos.x, pos.y, pos.z) - hit_sample;
          float drbf = length(hit_normal) * inv_fixed_radius_scale;
          drbf = fmaxf(0.f, fminf(1.f, wScale * pos.w * exp(-drbf*drbf)));
          float4 color_sample = tf(drbf, trbf * redshiftScale, tf_type);

          float alpha = color_sample.w * opacity;
          float alpha_1msa = alpha * (1.0 - result_alpha);
          result += make_float3(color_sample.x, color_sample.y, color_sample.z) * alpha_1msa;
          result_alpha += alpha_1msa;
        }
      }

      tbuffer += slab_spacing;
    }

  }

  //write to frame buffer
  float4 acc_val =  make_float4(result, 0.f);
  output_buffer[launch_index] = make_color( make_float3( acc_val ) );
  accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
}

