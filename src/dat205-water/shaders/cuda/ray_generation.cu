#include "hip/hip_runtime.h"
#include "common.cuh"

// The 2D color (RGBA32F) buffer we will render our result to.
rtBuffer<float4,  2> output_buffer;

// Root object of the scene.
rtDeclareVariable(rtObject, root, , );

// Output buffer/screen dimensionality and current pixel index.
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim  , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, ); // NOTE: launchIndex = (0, 0) is the bot-left corner (thus, matches OpenGL's texture origin) of the buffer/screen.

// Camera.
rtDeclareVariable(float3, camera_pos    , , );
rtDeclareVariable(float3, camera_right  , , );
rtDeclareVariable(float3, camera_up     , , );
rtDeclareVariable(float3, camera_forward, , );

// Entry point for this ray tracing kernel.
RT_PROGRAM void ray_generation() {

  const float2 pixel = make_float2(launch_index);
  const float2 pixel_center = pixel + make_float2(0.5f);

  // Shoots jittered samples to approximate the true pixel radiance.
  const unsigned int n_samples = 1;
  float3 tot_radiance = make_float3(0.0f);

  for (int i = 0; i < n_samples; i++) {

    // Generate a small offset from the pixel center.
    unsigned int seed = launch_dim.x * launch_index.x + launch_dim.y * launch_index.y + i;
    float2 subpixel_jitter = make_float2(rnd(seed) - 0.5f, rnd(seed) - 0.5f);
    subpixel_jitter *= 0.5f; // Looks a bit better with this when only using a single sample.

    // Derive the ray direction for the sampled subpixel.
    const float2 screen = make_float2(launch_dim);
    const float2 ndc = ((pixel_center + subpixel_jitter) / screen) * 2.0f - 1.0f; // [-1, 1] Normalized Device Coordinates
    const float3 direction = optix::normalize(ndc.x * camera_right + ndc.y * camera_up + camera_forward); // NOTE: the direction must be normalized.

    // Setup ray from origin, towards direction, using ray type 0, and test the interval between 0.0f and RT_DEFAULT_MAX for intersections.
    optix::Ray ray = optix::make_Ray(camera_pos, direction, 0, 0.0f, RT_DEFAULT_MAX);

    // Shoot the ray, storing the result in payload.
    RayPayload payload;
    payload.radiance        = make_float3(0.0f);
    payload.importance      = 1.0f;
    payload.recursion_depth = 0;

    rtTrace(root, ray, payload);

    tot_radiance += payload.radiance;
  }

  // Write result to output buffer.
  output_buffer[launch_index] = make_float4(tot_radiance / n_samples, 1.0f);
}