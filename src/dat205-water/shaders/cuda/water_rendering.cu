#include "hip/hip_runtime.h"
#include "common.cuh"

// The current ray and its payload.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay , );
rtDeclareVariable(RayPayload, payload, rtPayload, );

// Simulated particles.
rtBuffer<Particle> particles_buffer;

// Current particle.
rtDeclareVariable(Particle, attr_particle, attribute PARTICLE, );

// RT_PROGRAM void any_hit() {
//   payload.radiance = make_float3(0.0, 0.0, 0.0f);
// }

RT_PROGRAM void closest_hit() {
  // const float speed = optix::length(attr_particle.velocity);
  // payload.radiance = make_float3(speed, speed, 1.0f);

  // payload.radiance = make_float3(0.0f, 0.0f, 1.0f);
  payload.radiance = attr_particle.velocity;
}

RT_PROGRAM void bounding_box(int primitive_index, float result[6]) {
  optix::Aabb *aabb = (optix::Aabb *) result;

  const float3 pos = particles_buffer[primitive_index].position;

  // Enclose the particle with a cube.
  aabb->m_min = pos - make_float3(PARTICLE_RADIUS);
  aabb->m_max = pos + make_float3(PARTICLE_RADIUS);
}

RT_PROGRAM void ray_intersection(int primitive_index) {

  // Ray: r(t) = o + td
  // Sphere: ||p - c|| = r

  // We solve: dot(r(t) - c, r(t) - c) - r^2 = 0
  //
  // , which results in: t^2 + 2t * dot(o - c, d) + dot(o - c, o - c) - r^2 = 0
  //
  // , whose solution for t is: t = - dot(o - c, d) \pm sqrt(dot(o - c, d) * dot(o - c, d) - dot(o - c, o - c) + r^2)

  const float3 o = ray.origin;
  const float3 d = ray.direction;
  const float3 c = particles_buffer[primitive_index].position;
  const float r = PARTICLE_RADIUS;

  // Compute only once
  const float oc_d = optix::dot(o - c, d);
  const float inside_root_term = oc_d * oc_d- optix::dot(o - c, o - c) + r * r;

  // Ensure that any (intersection) solution exists.
  if (0.0f <= inside_root_term) {

    // Compute only once
    const float sqrt_term = sqrtf(inside_root_term);

    // Consider the two (usually different) possible solutions.
    const float t1 = -oc_d - sqrt_term;

    // Determine whether the reported hit distance is within the valid interval associated with the ray.
    if (rtPotentialIntersection(t1)) {
      attr_particle = particles_buffer[primitive_index];
      rtReportIntersection(0);
      return;
    }

    // NOTE: We might want to ignore the second root for performance reasons.
    const float t2 = -oc_d + sqrt_term;
    if (rtPotentialIntersection(t2)) {
      rtReportIntersection(0);
    }
  }
}