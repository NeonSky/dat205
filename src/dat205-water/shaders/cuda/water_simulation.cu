#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );

rtDeclareVariable(float, dt, , ); // Delta time
rtDeclareVariable(float, g , , ); // Gravity acceleration

// Simulated particles.
rtBuffer<Particle> particles_buffer;


RT_PROGRAM void update() {
    // rtPrintf("Launch index %d \n", launch_index);

    particles_buffer[launch_index].position += make_float3(dt);

    const float ratio = (float)launch_index / (float)launch_dim;
    particles_buffer[launch_index].velocity = make_float3(ratio);
}