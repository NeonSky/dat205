#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );

rtDeclareVariable(float, dt    , , ); // Delta time [s]
rtDeclareVariable(float, g     , , ); // Gravity acceleration [m/s^2]

rtDeclareVariable(float, cell_size, , ); // [m]
rtDeclareVariable(float, support_radius, , ); // [m]
rtDeclareVariable(float, particle_radius, , ); // [m]

rtDeclareVariable(float, particle_mass, , );// [kg]
rtDeclareVariable(float, rest_density, , ); // [kg / m^3]
rtDeclareVariable(float, gass_stiffness, , ); // [Pa * m^3 / kg]
rtDeclareVariable(float, viscosity, , ); // [Pa * s]
rtDeclareVariable(float, l_threshold, , ); // []
rtDeclareVariable(float, surface_tension, , ); // [N / m]
rtDeclareVariable(float, restitution, , ); // []

rtDeclareVariable(float, y_min , , ); // The floor's y-level
rtDeclareVariable(float, x_min , , ); // Left wall
rtDeclareVariable(float, x_max , , ); // Right wall
rtDeclareVariable(float, z_min , , ); // Far wall
rtDeclareVariable(float, z_max , , ); // Near wall

rtBuffer<HashCell> hash_table;

// Simulated particles.
rtBuffer<Particle> particles_buffer;


// eq 5.1, 5.2, 5.3
RT_FUNCTION uint hash(int3 pos) {
  // Primes
  static const int p1 = 73856093;
  static const int p2 = 19349663;
  static const int p3 = 83492791;

  // Hash
  return ((pos.x * p1) ^ (pos.y * p2) ^ (pos.z * p3)) % hash_table.size();
}

RT_PROGRAM void reset_nearest_neighbors() {
  Particle& p = particles_buffer[launch_index];
  HashCell& cell = hash_table[p.prev_hash_cell_index];
  cell[0] = 0; // only necessary to reset the count
}

RT_PROGRAM void update_nearest_neighbors() {
  Particle& p = particles_buffer[launch_index];

  int3 cell_position = make_int3(p.position / cell_size);
  uint cell_index = hash(cell_position);
  HashCell& cell = hash_table[cell_index];
  p.prev_hash_cell_index = cell_index;

  // Increase the particle count counter
  uint prev_particle_count = atomicAdd(&cell[0], 1);

  // Were we already at max?
  if (prev_particle_count >= HASH_CELL_SIZE-1) {
    // Revert
    atomicMin(&cell[0], HASH_CELL_SIZE-1);
  }
  // Otherwise, we have now reserved a spot (prev_particle_count) in the hash cell
  else {
    cell[1 + prev_particle_count] = launch_index;
  }
}

RT_FUNCTION void nearest_neighbor_search(Particle& p,
                                         unsigned int& nn_count,
                                         unsigned int* nn) {

  int3 center_cell_position = make_int3(p.position / cell_size);

  // Search for neighbors in the 3x3x3 grid of cells that is centered on this particle's cell.
  for (int x = -1; x <= 1; x++) {
    for (int y = -1; y <= 1; y++) {
      for (int z = -1; z <= 1; z++) {
        int3 cell_position = center_cell_position + make_int3(x, y, z);
        uint cell_index = hash(cell_position);
        HashCell& cell = hash_table[cell_index];

        uint particles_in_cell = cell[0];
        for (int i = 1; i <= particles_in_cell; i++) {
          if (cell[i] != launch_index) {
            nn[nn_count] = cell[i];
            nn_count += 1;
          }
        }
      }
    }
  }
}

// eq 4.3
RT_FUNCTION float poly6_kernel(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return (315.0f / (64.0f * M_PIf * powf(support_radius, 9.0f))) * powf(powf(support_radius, 2.0f) - powf(distance, 2.0f), 3.0f);
  }
}

// eq 4.4
RT_FUNCTION float3 poly6_kernel_gradient(float3 dist_vec) {
  float distance = optix::length(dist_vec);
  if (distance >= support_radius) {
    return make_float3(0.0f);
  } else {
    return -(945.0f / (32.0f * M_PIf * powf(support_radius, 9.0f))) * dist_vec * powf(powf(support_radius, 2.0f) - powf(distance, 2.0f), 2.0f);
  }
}

// eq 4.5
RT_FUNCTION float poly6_kernel_laplacian(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return -(945.0f / (32.0f * M_PIf * powf(support_radius, 9.0f))) * (powf(support_radius, 2.0f) - powf(distance, 2.0f)) * (3.0f * powf(support_radius, 2.0f) - 7.0f * powf(distance, 2.0f));
  }
}

// eq 4.6
RT_FUNCTION void update_density(Particle& p,
                                unsigned int nn_count,
                                unsigned int* nn) {

  // Density from this particle alone
  float density = particle_mass * poly6_kernel(0.0f);

  // Density from neighbors
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float distance = optix::length(p.position - pi.position);

    density += particle_mass * poly6_kernel(distance);
  }

  p.density = density;
}

// eq 4.12
RT_FUNCTION void update_pressure(Particle& p,
                                unsigned int nn_count,
                                unsigned int* nn) {
  p.pressure = gass_stiffness * (p.density - rest_density);
}

RT_PROGRAM void update_particles_data() {
  Particle& p = particles_buffer[launch_index];

  // Find nearest neighbors
  unsigned int nn_count = 0;
  unsigned int nn[3 * 3 * 3 * HASH_CELL_SIZE];
  nearest_neighbor_search(p, nn_count, nn);

  // Update density and pressure for each particle
  update_density(p, nn_count, nn);
  update_pressure(p, nn_count, nn);
}

// eq 4.14
RT_FUNCTION float3 pressure_kernel_gradient(float3 dist_vec) {
  float distance = optix::length(dist_vec);
  if (distance >= support_radius) {
    return make_float3(0.0f);
  } else if (distance < 1e-5) {
    // We'll treat this case as the particles being in the same position.
    // Thus, they won't affect each other in any direction.
    // This is mostly to avoid division by zero.
    // return make_float3(0.0f);
    return -(45.0f / (M_PIf * powf(support_radius, 6.0f))) * optix::normalize(make_float3(1.0f)) * powf(support_radius - distance, 2.0f);
  } else {
    return -(45.0f / (M_PIf * powf(support_radius, 6.0f))) * (dist_vec / distance) * powf(support_radius - distance, 2.0f);
  }
}

// eq 4.10
RT_FUNCTION float3 pressure_force(Particle& p,
                                  unsigned int nn_count,
                                  unsigned int* nn) {
  float3 force = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float3 dist_vec = p.position - pi.position;

    force += particle_mass * (p.pressure / powf(p.density, 2.0f) + pi.pressure / powf(pi.density, 2.0f)) * pressure_kernel_gradient(dist_vec);
  }
  force *= -1.0f * p.density;
  return force;
}

// eq 4.22
RT_FUNCTION float viscosity_kernel_laplacian(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return (45.0f / (M_PIf * powf(support_radius, 6.0f))) * (support_radius - distance);
  }
}

// eq 4.17
RT_FUNCTION float3 viscosity_force(Particle& p,
                                   unsigned int nn_count,
                                   unsigned int* nn) {
  float3 force = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];

    force += (pi.velocity - p.velocity) * (particle_mass / pi.density) * viscosity_kernel_laplacian(optix::length(pi.position - p.position));
  }
  force *= viscosity;
  return force;
}

// eq 4.24
RT_FUNCTION float3 gravity_force(float particle_density) {
  return particle_density * make_float3(0.0f, g, 0.0f);
}

RT_FUNCTION float3 surface_tension_force(Particle& p,
                                         unsigned int nn_count,
                                         unsigned int* nn) {

  // eq 4.28
  float3 inward_surface_normal = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];

    inward_surface_normal += (particle_mass / pi.density) * poly6_kernel_gradient(p.position - pi.position);
  }

  float normal_dist = optix::length(inward_surface_normal);
  if (normal_dist < l_threshold) {
    return make_float3(0.0f);
  }

  // eq 4.26
  float laplacian = (particle_mass / p.density) * poly6_kernel_laplacian(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float distance = optix::length(p.position - pi.position);

    laplacian += (particle_mass / pi.density) * poly6_kernel_laplacian(distance);
  }

  float3 force = -surface_tension * laplacian * (inward_surface_normal / normal_dist);

  return force;
}

RT_FUNCTION void euler_cromer(Particle& p, float3 force) {
    float3 acceleration = force / p.density; // eq 4.2
    p.velocity += dt * acceleration;
    p.position += dt * p.velocity;
}

RT_FUNCTION void collision_detection(Particle& p) {

  // Early exit
  if (x_min <= p.position.x && p.position.x <= x_max &&
      y_min <= p.position.y &&
      z_min <= p.position.z && p.position.z <= z_max) {
    return;
  }

  float3 contact_point = p.position;
  contact_point.x = min(x_max, max(x_min, p.position.x));
  contact_point.y = max(y_min, p.position.y);
  contact_point.z = min(z_max, max(z_min, p.position.z));

  char maxComponent = 'y';
  float maxDepth    = y_min - p.position.y;

  if (maxDepth < x_min - p.position.x) {
      maxComponent = 'x';
      maxDepth = x_min - p.position.x;
  } else if (maxDepth < p.position.x - x_max) {
      maxComponent = 'x';
      maxDepth = p.position.x - x_max;
  }

  if (maxDepth < z_min - p.position.z) {
      maxComponent = 'z';
      maxDepth = z_min - p.position.z;
  } else if (maxDepth < p.position.z - z_max) {
      maxComponent = 'z';
      maxDepth = p.position.z - z_max;
  }

  float3 surface_normal = make_float3(0.0f);
  switch (maxComponent) {
    case 'x':
      if (p.position.x < x_min) {
          surface_normal = make_float3(1.0f,  0.0f,  0.0f);
      }
      else if (p.position.x > x_max) {
          surface_normal = make_float3(-1.0f,  0.0f,  0.0f);
      }
      break;
    case 'y':
      if (p.position.y < y_min) {
          surface_normal = make_float3(0.0f,  1.0f,  0.0f);
      }
      break;
    case 'z':
      if (p.position.z < z_min) {
          surface_normal = make_float3(0.0f,  0.0f,  1.0f);
      }
      else if (p.position.z > z_max) {
          surface_normal = make_float3(0.0f,  0.0f, -1.0f);
      }
      break;
  }

  // eq 4.58
  float penetration_depth = optix::length(p.position - contact_point);
  p.velocity = p.velocity - (1.0f + restitution * penetration_depth / (dt * optix::length(p.velocity))) * optix::dot(p.velocity, surface_normal) * surface_normal;
  p.position = contact_point;
}

// p54
RT_PROGRAM void update() {
    Particle& p = particles_buffer[launch_index];

    // Find nearest neighbors
    unsigned int nn_count = 0;
    unsigned int nn[3 * 3 * 3 * HASH_CELL_SIZE];
    nearest_neighbor_search(p, nn_count, nn);

    float3 tot_force = make_float3(0.0f);

    // Internal forces
    tot_force += pressure_force(p, nn_count, nn);
    tot_force += viscosity_force(p, nn_count, nn);

    // External forces
    tot_force += gravity_force(p.density);
    tot_force += surface_tension_force(p, nn_count, nn);

    // Integrate forces over time
    euler_cromer(p, tot_force);

    // Handle potential collisions
    collision_detection(p);
}