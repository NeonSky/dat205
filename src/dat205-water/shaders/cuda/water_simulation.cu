#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );

rtDeclareVariable(float, dt    , , ); // Delta time
rtDeclareVariable(float, g     , , ); // Gravity acceleration
rtDeclareVariable(float, y_min , , ); // The floor's y-level
rtDeclareVariable(float, x_min , , ); // Left wall
rtDeclareVariable(float, x_max , , ); // Right wall
rtDeclareVariable(float, z_min , , ); // Far wall
rtDeclareVariable(float, z_max , , ); // Near wall

// Simulated particles.
rtBuffer<Particle> particles_buffer;


RT_PROGRAM void update() {
    // rtPrintf("Launch index %d \n", launch_index);
    Particle& p = particles_buffer[launch_index];

    p.position += dt * p.velocity;
    p.velocity.y += dt * g;

    const float radius_of_concern = 1.0f;
    for (unsigned int i = 0; i < launch_dim; i++) {
      if (i != launch_index) {
        float3 away_vec = p.position - particles_buffer[i].position;
        float dist      = optix::length(away_vec);

        if (radius_of_concern < dist) {
          continue;
        }

        p.velocity += dt * away_vec / (dist * dist * M_PIf);
      }
    }

    if (p.position.x <= x_min + PARTICLE_RADIUS) {
      p.position.x = x_min + PARTICLE_RADIUS;
      p.velocity.x *= -0.8f;
    }
    else if (p.position.x >= x_max - PARTICLE_RADIUS) {
      p.position.x = x_max - PARTICLE_RADIUS;
      p.velocity.x *= -0.8f;
    }

    if (p.position.z <= z_min + PARTICLE_RADIUS) {
      p.position.z = z_min + PARTICLE_RADIUS;
      p.velocity.z *= -0.8f;
    }
    else if (p.position.z >= z_max - PARTICLE_RADIUS) {
      p.position.z = z_max - PARTICLE_RADIUS;
      p.velocity.z *= -0.8f;
    }

    if (p.position.y <= y_min + PARTICLE_RADIUS) {
      p.position.y = y_min + PARTICLE_RADIUS;
      // p.velocity.y = 0.0f;
      p.velocity.y *= -0.7f;
    }
}