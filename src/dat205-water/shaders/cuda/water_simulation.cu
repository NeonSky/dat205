#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );

rtDeclareVariable(float, dt    , , ); // Delta time [s]
rtDeclareVariable(float, g     , , ); // Gravity acceleration [m/s^2]

rtDeclareVariable(float, cell_size, , ); // [m]
rtDeclareVariable(float, support_radius, , ); // [m]

rtDeclareVariable(float, particle_mass, , );// [kg]
rtDeclareVariable(float, rest_density, , ); // [kg / m^3]
rtDeclareVariable(float, gass_stiffness, , ); // [Pa * m^3 / kg]
rtDeclareVariable(float, viscosity, , ); // [Pa * s]

rtDeclareVariable(float, y_min , , ); // The floor's y-level
rtDeclareVariable(float, x_min , , ); // Left wall
rtDeclareVariable(float, x_max , , ); // Right wall
rtDeclareVariable(float, z_min , , ); // Far wall
rtDeclareVariable(float, z_max , , ); // Near wall

rtBuffer<HashCell> hash_table;

// Simulated particles.
rtBuffer<Particle> particles_buffer;


// eq 5.1, 5.2, 5.3
RT_FUNCTION uint hash(int3 pos) {
  // Primes
  static const int p1 = 73856093;
  static const int p2 = 19349663;
  static const int p3 = 83492791;

  // Hash
  return ((pos.x * p1) ^ (pos.y * p2) ^ (pos.z * p3)) % hash_table.size();
}

// TODO: this might be faster to do on the CPU
RT_PROGRAM void update_nearest_neighbors() {
  // rtPrintf("\n");
  for (uint i = 0; i < particles_buffer.size(); i++) {
    Particle& p = particles_buffer[i];

    int3 cell_position = make_int3(p.position / cell_size);
    // rtPrintf("(%d, %d, %d) \n", cell_position.x, cell_position.y, cell_position.z);
    HashCell& cell = hash_table[hash(cell_position)];
    // rtPrintf("(%u)\n", hash(cell_position));

    uint particles_in_cell = cell[0];
    if (particles_in_cell < HASH_CELL_SIZE-1) {
      cell[0] += 1;
      cell[particles_in_cell+1] = i;
    }
  }
}

RT_FUNCTION void nearest_neighbor_search(Particle& p,
                                         unsigned int& nn_count,
                                         unsigned int* nn) {

  int3 center_cell_position = make_int3(p.position / cell_size);

  // Search for neighbors in the 3x3x3 grid of cells that is centered on this particle's cell.
  for (int x = -1; x <= 1; x++) {
    for (int y = -1; y <= 1; y++) {
      for (int z = -1; z <= 1; z++) {
        int3 cell_position = center_cell_position + make_int3(x, y, z);
        HashCell& cell = hash_table[hash(cell_position)];

        uint particles_in_cell = cell[0];
        for (int i = 1; i <= particles_in_cell; i++) {
          if (cell[i] != launch_index) {
            nn[nn_count] = cell[i];
            nn_count += 1;
          }
        }
      }
    }
  }
}

// eq 4.3
RT_FUNCTION float poly6_kernel(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return (315.0f / (64.0f * M_PIf * powf(support_radius, 9.0f))) * powf(powf(support_radius, 2.0f) - powf(distance, 2.0f), 3.0f);
  }
}

// eq 4.4
// RT_FUNCTION float poly6_kernel_gradient(float distance, float support_radius) {
//   return -(945.0f / (32.0f * M_PIf * powf(support_radius, 9.0f))) * dist_vector * powf(powf(support_radius, 2.0f) - powf(distance, 2.0f), 2.0f)
// }

// eq 4.6
RT_FUNCTION void update_density(Particle& p,
                                unsigned int nn_count,
                                unsigned int* nn) {

  // Density from this particle alone
  float density = particle_mass * poly6_kernel(0.0f);

  // Density from neighbors
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float distance = optix::length(p.position - pi.position);

    density += particle_mass * poly6_kernel(distance);
  }

  p.density = density;
}

// eq 4.12
RT_FUNCTION void update_pressure(Particle& p,
                                unsigned int nn_count,
                                unsigned int* nn) {
  p.pressure = gass_stiffness * (p.density - rest_density);
}

RT_PROGRAM void update_particles_data() {
  Particle& p = particles_buffer[launch_index];

  // Find nearest neighbors
  unsigned int nn_count = 0;
  unsigned int nn[3 * 3 * 3 * HASH_CELL_SIZE];
  nearest_neighbor_search(p, nn_count, nn);

  // Update density and pressure for each particle
  update_density(p, nn_count, nn);
  update_pressure(p, nn_count, nn);
}

// eq 4.14
RT_FUNCTION float3 pressure_kernel_gradient(float3 dist_vec) {
  float distance = optix::length(dist_vec);
  if (distance >= support_radius) {
    return make_float3(0.0f);
  } else if (distance < 1e-5) {
    // We'll treat this case as the particles being in the same position.
    // Thus, they won't affect each other in any direction.
    // This is mostly to avoid division by zero.
    return make_float3(0.0f);
  } else {
    return -(45.0f / (M_PIf * powf(support_radius, 6.0f))) * (dist_vec / distance) * powf(support_radius - distance, 2.0f);
  }
}

// eq 4.11
RT_FUNCTION float3 pressure_force(Particle& p,
                                  unsigned int nn_count,
                                  unsigned int* nn) {
  float3 force = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float3 dist_vec = p.position - pi.position;

    force += particle_mass * ((p.pressure + pi.pressure) / (2.0f * pi.density)) * pressure_kernel_gradient(dist_vec);
  }
  force *= -1.0f;
  return force;
}

// eq 4.22
RT_FUNCTION float viscosity_kernel_laplacian(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return (45.0f / (M_PIf * powf(support_radius, 6.0f))) * (support_radius - distance);
  }
}

// eq 4.17
RT_FUNCTION float3 viscosity_force(Particle& p,
                                   unsigned int nn_count,
                                   unsigned int* nn) {
  float3 force = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];

    force += (pi.velocity - p.velocity) * (particle_mass / pi.density) * viscosity_kernel_laplacian(optix::length(pi.position - p.position));
  }
  force *= viscosity;
  return force;
}

// eq 4.24
RT_FUNCTION float3 gravity_force(float particle_density) {
  return particle_density * make_float3(0.0f, g, 0.0f);
}

RT_FUNCTION void euler_cromer(Particle& p, float3 force) {
    float3 acceleration = force / p.density; // eq 4.2
    p.velocity += dt * acceleration;
    p.position += dt * p.velocity;
}

// p54
RT_PROGRAM void update() {
    Particle& p = particles_buffer[launch_index];

    // Find nearest neighbors
    unsigned int nn_count = 0;
    unsigned int nn[3 * 3 * 3 * HASH_CELL_SIZE];
    nearest_neighbor_search(p, nn_count, nn);

    float3 tot_force = make_float3(0.0f);

    // Internal forces
    tot_force += pressure_force(p, nn_count, nn);
    tot_force += viscosity_force(p, nn_count, nn);

    // External forces
    tot_force += gravity_force(p.density);

    // Integrate forces over time
    euler_cromer(p, tot_force);

    // TODO: Remove
    if (p.position.x <= x_min + PARTICLE_RADIUS) {
      p.position.x = x_min + PARTICLE_RADIUS;
      p.velocity.x *= -0.5f;
    }
    else if (p.position.x >= x_max - PARTICLE_RADIUS) {
      p.position.x = x_max - PARTICLE_RADIUS;
      p.velocity.x *= -0.5f;
    }

    if (p.position.z <= z_min + PARTICLE_RADIUS) {
      p.position.z = z_min + PARTICLE_RADIUS;
      p.velocity.z *= -0.5f;
    }
    else if (p.position.z >= z_max - PARTICLE_RADIUS) {
      p.position.z = z_max - PARTICLE_RADIUS;
      p.velocity.z *= -0.5f;
    }

    if (p.position.y <= y_min + PARTICLE_RADIUS) {
      p.position.y = y_min + PARTICLE_RADIUS;
      p.velocity.y = 0.0f;
      // p.velocity.y *= -0.7f;
    }
}