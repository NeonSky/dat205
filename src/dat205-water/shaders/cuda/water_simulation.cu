#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );

rtDeclareVariable(float, dt    , , ); // Delta time [s]
rtDeclareVariable(float, g     , , ); // Gravity acceleration [m/s^2]

rtDeclareVariable(float, cell_size, , ); // [m]
rtDeclareVariable(float, support_radius, , ); // [m]

rtDeclareVariable(float, particle_mass, , );// [kg]
rtDeclareVariable(float, rest_density, , ); // [kg / m^3]
rtDeclareVariable(float, gass_stiffness, , ); // [Pa * m^3 / kg]
rtDeclareVariable(float, viscosity, , ); // [Pa * s]
rtDeclareVariable(float, l_threshold, , ); // []
rtDeclareVariable(float, surface_tension, , ); // [N / m]

rtDeclareVariable(float, y_min , , ); // The floor's y-level
rtDeclareVariable(float, x_min , , ); // Left wall
rtDeclareVariable(float, x_max , , ); // Right wall
rtDeclareVariable(float, z_min , , ); // Far wall
rtDeclareVariable(float, z_max , , ); // Near wall

rtBuffer<HashCell> hash_table;

// Simulated particles.
rtBuffer<Particle> particles_buffer;


// eq 5.1, 5.2, 5.3
RT_FUNCTION uint hash(int3 pos) {
  // Primes
  static const int p1 = 73856093;
  static const int p2 = 19349663;
  static const int p3 = 83492791;

  // Hash
  return ((pos.x * p1) ^ (pos.y * p2) ^ (pos.z * p3)) % hash_table.size();
}

RT_PROGRAM void reset_nearest_neighbors() {
  Particle& p = particles_buffer[launch_index];
  HashCell& cell = hash_table[p.prev_hash_cell_index];
  cell[0] = 0; // only necessary to reset the count
}

RT_PROGRAM void update_nearest_neighbors() {
  Particle& p = particles_buffer[launch_index];

  int3 cell_position = make_int3(p.position / cell_size);
  uint cell_index = hash(cell_position);
  HashCell& cell = hash_table[cell_index];
  p.prev_hash_cell_index = cell_index;

  uint particles_in_cell = cell[0];
  if (particles_in_cell < HASH_CELL_SIZE-1) {
    atomicAdd(&cell[0], 1);
    cell[particles_in_cell+1] = launch_index;
  }
}

RT_FUNCTION void nearest_neighbor_search(Particle& p,
                                         unsigned int& nn_count,
                                         unsigned int* nn) {

  int3 center_cell_position = make_int3(p.position / cell_size);

  // Search for neighbors in the 3x3x3 grid of cells that is centered on this particle's cell.
  for (int x = -1; x <= 1; x++) {
    for (int y = -1; y <= 1; y++) {
      for (int z = -1; z <= 1; z++) {
        int3 cell_position = center_cell_position + make_int3(x, y, z);
        HashCell& cell = hash_table[hash(cell_position)];

        uint particles_in_cell = cell[0];
        for (int i = 1; i <= particles_in_cell; i++) {
          if (cell[i] != launch_index) {
            nn[nn_count] = cell[i];
            nn_count += 1;
          }
        }
      }
    }
  }
}

// eq 4.3
RT_FUNCTION float poly6_kernel(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return (315.0f / (64.0f * M_PIf * powf(support_radius, 9.0f))) * powf(powf(support_radius, 2.0f) - powf(distance, 2.0f), 3.0f);
  }
}

// eq 4.4
RT_FUNCTION float3 poly6_kernel_gradient(float3 dist_vec) {
  float distance = optix::length(dist_vec);
  if (distance >= support_radius) {
    return make_float3(0.0f);
  } else {
    return -(945.0f / (32.0f * M_PIf * powf(support_radius, 9.0f))) * dist_vec * powf(powf(support_radius, 2.0f) - powf(distance, 2.0f), 2.0f);
  }
}

// eq 4.5
RT_FUNCTION float poly6_kernel_laplacian(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return -(945.0f / (32.0f * M_PIf * powf(support_radius, 9.0f))) * (powf(support_radius, 2.0f) - powf(distance, 2.0f)) * (3.0f * powf(support_radius, 2.0f) - 7.0f * powf(distance, 2.0f));
  }
}

// eq 4.6
RT_FUNCTION void update_density(Particle& p,
                                unsigned int nn_count,
                                unsigned int* nn) {

  // Density from this particle alone
  float density = particle_mass * poly6_kernel(0.0f);

  // Density from neighbors
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float distance = optix::length(p.position - pi.position);

    density += particle_mass * poly6_kernel(distance);
  }

  p.density = density;
}

// eq 4.12
RT_FUNCTION void update_pressure(Particle& p,
                                unsigned int nn_count,
                                unsigned int* nn) {
  p.pressure = gass_stiffness * (p.density - rest_density);
}

RT_PROGRAM void update_particles_data() {
  Particle& p = particles_buffer[launch_index];

  // Find nearest neighbors
  unsigned int nn_count = 0;
  unsigned int nn[3 * 3 * 3 * HASH_CELL_SIZE];
  nearest_neighbor_search(p, nn_count, nn);

  // Update density and pressure for each particle
  update_density(p, nn_count, nn);
  update_pressure(p, nn_count, nn);
}

// eq 4.14
RT_FUNCTION float3 pressure_kernel_gradient(float3 dist_vec) {
  float distance = optix::length(dist_vec);
  if (distance >= support_radius) {
    return make_float3(0.0f);
  } else if (distance < 1e-5) {
    // We'll treat this case as the particles being in the same position.
    // Thus, they won't affect each other in any direction.
    // This is mostly to avoid division by zero.
    return make_float3(0.0f);
  } else {
    return -(45.0f / (M_PIf * powf(support_radius, 6.0f))) * (dist_vec / distance) * powf(support_radius - distance, 2.0f);
  }
}

// eq 4.11
RT_FUNCTION float3 pressure_force(Particle& p,
                                  unsigned int nn_count,
                                  unsigned int* nn) {
  float3 force = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float3 dist_vec = p.position - pi.position;

    force += particle_mass * ((p.pressure + pi.pressure) / (2.0f * pi.density)) * pressure_kernel_gradient(dist_vec);
  }
  force *= -1.0f;
  return force;
}

// eq 4.22
RT_FUNCTION float viscosity_kernel_laplacian(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return (45.0f / (M_PIf * powf(support_radius, 6.0f))) * (support_radius - distance);
  }
}

// eq 4.17
RT_FUNCTION float3 viscosity_force(Particle& p,
                                   unsigned int nn_count,
                                   unsigned int* nn) {
  float3 force = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];

    force += (pi.velocity - p.velocity) * (particle_mass / pi.density) * viscosity_kernel_laplacian(optix::length(pi.position - p.position));
  }
  force *= viscosity;
  return force;
}

// eq 4.24
RT_FUNCTION float3 gravity_force(float particle_density) {
  return particle_density * make_float3(0.0f, g, 0.0f);
}

RT_FUNCTION float3 surface_tension_force(Particle& p,
                                         unsigned int nn_count,
                                         unsigned int* nn) {

  // eq 4.28
  float3 inward_surface_normal = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float distance = optix::length(p.position - pi.position);

    inward_surface_normal += (particle_mass / pi.density) * poly6_kernel_gradient(p.position - pi.position);
  }

  float normal_dist = optix::length(inward_surface_normal);
  if (normal_dist < l_threshold) {
    return make_float3(0.0f);
  }

  // eq 4.26
  float laplacian = 0.0f;
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float distance = optix::length(p.position - pi.position);

    laplacian += (particle_mass / pi.density) * poly6_kernel_laplacian(distance);
  }
  float3 force = -surface_tension * laplacian * inward_surface_normal / normal_dist;

  return force;
}

RT_FUNCTION void euler_cromer(Particle& p, float3 force) {
    float3 acceleration = force / p.density; // eq 4.2
    p.velocity += dt * acceleration;
    p.position += dt * p.velocity;
}

// p54
RT_PROGRAM void update() {
    Particle& p = particles_buffer[launch_index];

    // Find nearest neighbors
    unsigned int nn_count = 0;
    unsigned int nn[3 * 3 * 3 * HASH_CELL_SIZE];
    nearest_neighbor_search(p, nn_count, nn);

    float3 tot_force = make_float3(0.0f);

    // Internal forces
    tot_force += pressure_force(p, nn_count, nn);
    tot_force += viscosity_force(p, nn_count, nn);

    // External forces
    tot_force += gravity_force(p.density);
    tot_force += surface_tension_force(p, nn_count, nn);

    // Integrate forces over time
    euler_cromer(p, tot_force);

    // TODO: Remove
    if (p.position.x <= x_min + PARTICLE_RADIUS) {
      p.position.x = x_min + PARTICLE_RADIUS;
      p.velocity.x *= -0.5f;
    }
    else if (p.position.x >= x_max - PARTICLE_RADIUS) {
      p.position.x = x_max - PARTICLE_RADIUS;
      p.velocity.x *= -0.5f;
    }

    if (p.position.z <= z_min + PARTICLE_RADIUS) {
      p.position.z = z_min + PARTICLE_RADIUS;
      p.velocity.z *= -0.5f;
    }
    else if (p.position.z >= z_max - PARTICLE_RADIUS) {
      p.position.z = z_max - PARTICLE_RADIUS;
      p.velocity.z *= -0.5f;
    }

    if (p.position.y <= y_min + PARTICLE_RADIUS) {
      p.position.y = y_min + PARTICLE_RADIUS;
      p.velocity.y = fmaxf(p.velocity.y, 0.0f);
      // p.velocity.y *= -0.7f;
    }
}