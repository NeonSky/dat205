#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );

rtDeclareVariable(float, dt    , , ); // Delta time
rtDeclareVariable(float, g     , , ); // Gravity acceleration
rtDeclareVariable(float, y_min , , ); // The floor's y-level
rtDeclareVariable(float, x_min , , ); // Left wall
rtDeclareVariable(float, x_max , , ); // Right wall
rtDeclareVariable(float, z_min , , ); // Far wall
rtDeclareVariable(float, z_max , , ); // Near wall

// Simulated particles.
rtBuffer<Particle> particles_buffer;


RT_PROGRAM void update() {
    // rtPrintf("Launch index %d \n", launch_index);
    Particle& p = particles_buffer[launch_index];

    p.position += dt * p.velocity;
    p.velocity.y += dt * g;

    if (p.position.x <= x_min + PARTICLE_RADIUS) {
      p.position.x = x_min + PARTICLE_RADIUS;
      p.velocity.x *= -1.0f;
    }
    else if (p.position.x >= x_max - PARTICLE_RADIUS) {
      p.position.x = x_max - PARTICLE_RADIUS;
      p.velocity.x *= -1.0f;
    }

    if (p.position.z <= z_min + PARTICLE_RADIUS) {
      p.position.z = z_min + PARTICLE_RADIUS;
      p.velocity.z *= -1.0f;
    }
    else if (p.position.z >= z_max - PARTICLE_RADIUS) {
      p.position.z = z_max - PARTICLE_RADIUS;
      p.velocity.z *= -1.0f;
    }

    if (p.position.y <= y_min + PARTICLE_RADIUS) {
      p.position.y = y_min + PARTICLE_RADIUS;
      // p.velocity.y = 0.0f;
      p.velocity.y *= -0.7f;
    }
}