#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );

rtDeclareVariable(float, dt, , ); // Delta time [s]
rtDeclareVariable(float, g , , ); // Gravity acceleration [m / s^2]

rtDeclareVariable(float, cell_size      , , ); // [m]
rtDeclareVariable(float, support_radius , , ); // [m]
rtDeclareVariable(float, particle_radius, , ); // [m]

rtDeclareVariable(float, particle_mass  , , ); // [kg]
rtDeclareVariable(float, rest_density   , , ); // [kg / m^3]
rtDeclareVariable(float, viscosity      , , ); // [Pa * s]
rtDeclareVariable(float, surface_tension, , ); // [N / m]
rtDeclareVariable(float, l_threshold    , , ); // []
rtDeclareVariable(float, gass_stiffness , , ); // [J]
rtDeclareVariable(float, restitution    , , ); // []

rtDeclareVariable(float, y_min, , ); // The floor's y-level
rtDeclareVariable(float, x_min, , ); // Left wall
rtDeclareVariable(float, x_max, , ); // Right wall
rtDeclareVariable(float, z_min, , ); // Far wall
rtDeclareVariable(float, z_max, , ); // Near wall

// A table/array/buffer of hash cells.
// In each cell we will store the particles that occupy that corresponding volume in space.
rtBuffer<HashCell> hash_table;

// Simulated particles.
rtBuffer<Particle> particles_buffer;


// Converts a discretized 3D position into a hash table index.
// We use this to decide where in the hash table to store each particle for neighbor detection.
//
// See: eq 5.1, 5.2, 5.3
RT_FUNCTION uint hash(int3 pos) {
  // Primes
  static const int p1 = 73856093;
  static const int p2 = 19349663;
  static const int p3 = 83492791;

  // Hash
  return ((pos.x * p1) ^ (pos.y * p2) ^ (pos.z * p3)) % hash_table.size();
}

// Resets each hash cell to contain 0 particles.
RT_PROGRAM void reset_nearest_neighbors() {
  Particle& p = particles_buffer[launch_index];

  // Fetch the previous cell we stored this particle in (default is 0).
  HashCell& cell = hash_table[p.prev_hash_cell_index];

  // Reset the particle count of that cell (idempotent operation).
  cell[0] = 0;
}

///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

// Updates each hash cell to contain the particles that occupy the corresponding physical space (voxel).
RT_PROGRAM void update_nearest_neighbors() {
  Particle& p = particles_buffer[launch_index];

  // Discretize the space into cells of size `cell_size` and compute particle `p`'s coordinates in this space.
  int3 cell_position = make_int3(p.position / cell_size);

  // Hash the discretized position to get its corresponding index in the hash table.
  uint cell_index = hash(cell_position);

  // Fetch the cell and remember its index (will be used the next time we reset the hash table).
  HashCell& cell = hash_table[cell_index];
  p.prev_hash_cell_index = cell_index;

  // Increase the particle occupaciation count.
  uint prev_particle_count = atomicAdd(&cell[0], 1);

  // Were we already at max before trying to add this new particle?
  // NOTE: first entry is for count so there are only `HASH_CELL_SIZE-1` particle slots.
  if (prev_particle_count >= HASH_CELL_SIZE-1) {
    // Revert the count to max (idempotent operation).
    atomicMin(&cell[0], HASH_CELL_SIZE-1);
  }

  // Otherwise, we have now reserved a spot (prev_particle_count) in the hash cell
  else {
    // Store particle `p`'s index in the particles buffer in this cell.
    // NOTE: we skip first entry since that's for the particle count.
    cell[1 + prev_particle_count] = launch_index;
  }
}

///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////

// Populates `nn` with the neighboring particles that are potentially within `support_radius` distance.
// Also updates `nn_count` to equal the amount of such particles.
RT_FUNCTION void nearest_neighbor_search(Particle& p,
                                         unsigned int& nn_count,
                                         unsigned int* nn) {

  int3 center_cell_position = make_int3(p.position / cell_size);

  // Search for neighbors in the 3x3x3 grid of cells that is centered on particle `p`.
  for (int x = -1; x <= 1; x++) {
    for (int y = -1; y <= 1; y++) {
      for (int z = -1; z <= 1; z++) {

        // Retrieve the corresponding hash cell.
        int3 cell_position = center_cell_position + make_int3(x, y, z);
        uint cell_index = hash(cell_position);
        HashCell& cell = hash_table[cell_index];

        // Iterate all particles in the cell.
        uint n_particles_in_cell = cell[0];
        for (int i = 1; i <= n_particles_in_cell; i++) {

          if (cell[i] != launch_index) {
            nn[nn_count] = cell[i];
            nn_count += 1;
          }
        }

      }
    }
  }
}

// The Poly 6 Smoothing Kernel
// Behaves similarly to a normal distribution but is quite fast to compute.
// NOTE: this is a normalized smoothing kernel, which means that \int W(|x - x_i|) dx = 1
// See eq 4.3
RT_FUNCTION float poly6_kernel(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return (315.0f / (64.0f * M_PIf * powf(support_radius, 9.0f))) * powf(powf(support_radius, 2.0f) - powf(distance, 2.0f), 3.0f);
  }
}

// The Gradient of the Poly 6 Smoothing Kernel
// See eq 4.4
RT_FUNCTION float3 poly6_kernel_gradient(float3 dist_vec) {
  float distance = optix::length(dist_vec);
  if (distance >= support_radius) {
    return make_float3(0.0f);
  } else {
    return -(945.0f / (32.0f * M_PIf * powf(support_radius, 9.0f))) * dist_vec * powf(powf(support_radius, 2.0f) - powf(distance, 2.0f), 2.0f);
  }
}

// The Laplacian of the Poly 6 Smoothing Kernel
// See eq 4.5
RT_FUNCTION float poly6_kernel_laplacian(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return -(945.0f / (32.0f * M_PIf * powf(support_radius, 9.0f))) * (powf(support_radius, 2.0f) - powf(distance, 2.0f)) * (3.0f * powf(support_radius, 2.0f) - 7.0f * powf(distance, 2.0f));
  }
}

// Evaluates the mass-density field at particle `p`.
// A smoothing kernel is used to ensure smooth continous densities across the fluid.
// See eq 4.6
RT_FUNCTION void update_density(Particle& p,
                                unsigned int nn_count,
                                unsigned int* nn) {

  // Density based on this particle alone.
  float density = particle_mass * poly6_kernel(0.0f);

  // Density from neighbors.
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float distance = optix::length(p.position - pi.position);

    density += particle_mass * poly6_kernel(distance);
  }

  p.density = density;
}

// Computes pressure based on density deviation from the fluid's rest density.
// This is a modification of the ideal gas law.
// eq 4.12
RT_FUNCTION void update_pressure(Particle& p) {
  // If density at `p` is relatively high, then so is pressure.
  // If density at `p` is relatively low, then so is pressure.
  p.pressure = gass_stiffness * (p.density - rest_density);
}

// Updates per-particle attributes based on neighboring particles.
RT_PROGRAM void update_particles_data() {
  Particle& p = particles_buffer[launch_index];

  // Find nearest neighbors.
  unsigned int nn_count = 0;
  unsigned int nn[3 * 3 * 3 * HASH_CELL_SIZE];
  nearest_neighbor_search(p, nn_count, nn);

  // Update density and pressure for each particle.
  update_density(p, nn_count, nn);
  update_pressure(p);
}

///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////
RT_FUNCTION float sign(float x) {
	return x > 0.0f ? 1.0f : -1.0f;
}

// The "spiky" kernel.
// This is used to prevent clusters when distance approaches 0.
// See eq 4.14
RT_FUNCTION float3 pressure_kernel_gradient(float3 dist_vec) {
  float distance = optix::length(dist_vec);
  if (distance >= support_radius) {
    return make_float3(0.0f);
  } else if (distance < 1e-3) {
    return -(45.0f / (M_PIf * powf(support_radius, 6.0f))) * make_float3(sign(dist_vec.x), sign(dist_vec.y), sign(dist_vec.z)) * powf(support_radius - distance, 2.0f);
  } else {
    return -(45.0f / (M_PIf * powf(support_radius, 6.0f))) * (dist_vec / distance) * powf(support_radius - distance, 2.0f);
  }
}

// eq 4.10
RT_FUNCTION float3 pressure_force(Particle& p,
                                  unsigned int nn_count,
                                  unsigned int* nn) {
  float3 force = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float3 dist_vec = p.position - pi.position;

    force += particle_mass * (p.pressure / powf(p.density, 2.0f) + pi.pressure / powf(pi.density, 2.0f)) * pressure_kernel_gradient(dist_vec);
  }
  force *= -1.0f * p.density;
  return force;
}

// eq 4.22
RT_FUNCTION float viscosity_kernel_laplacian(float distance) {
  if (distance >= support_radius) {
    return 0.0f;
  } else {
    return (45.0f / (M_PIf * powf(support_radius, 6.0f))) * (support_radius - distance);
  }
}

// eq 4.17
RT_FUNCTION float3 viscosity_force(Particle& p,
                                   unsigned int nn_count,
                                   unsigned int* nn) {
  float3 force = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];

    force += (pi.velocity - p.velocity) * (particle_mass / pi.density) * viscosity_kernel_laplacian(optix::length(pi.position - p.position));
  }
  force *= viscosity;
  return force;
}

// eq 4.24
RT_FUNCTION float3 gravity_force(float particle_density) {
  return particle_density * make_float3(0.0f, g, 0.0f);
}

RT_FUNCTION float3 surface_tension_force(Particle& p,
                                         unsigned int nn_count,
                                         unsigned int* nn) {

  // eq 4.28
  float3 inward_surface_normal = make_float3(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];

    inward_surface_normal += (particle_mass / pi.density) * poly6_kernel_gradient(p.position - pi.position);
  }

  float normal_dist = optix::length(inward_surface_normal);
  if (normal_dist < l_threshold) {
    return make_float3(0.0f);
  }

  // eq 4.26
  float laplacian = (particle_mass / p.density) * poly6_kernel_laplacian(0.0f);
  for (int i = 0; i < nn_count; i++) {
    Particle& pi = particles_buffer[nn[i]];
    float distance = optix::length(p.position - pi.position);

    laplacian += (particle_mass / pi.density) * poly6_kernel_laplacian(distance);
  }

  float3 force = -surface_tension * laplacian * (inward_surface_normal / normal_dist);

  return force;
}

RT_FUNCTION void euler_cromer(Particle& p, float3 force) {
    float3 acceleration = force / p.density; // eq 4.2
    p.velocity += dt * acceleration;
    p.position += dt * p.velocity;
}

RT_FUNCTION void collision_detection(Particle& p) {

  // Early exit
  if (x_min <= p.position.x && p.position.x <= x_max &&
      y_min <= p.position.y &&
      z_min <= p.position.z && p.position.z <= z_max) {
    return;
  }

  float3 contact_point = p.position;
  contact_point.x = min(x_max, max(x_min, p.position.x));
  contact_point.y = max(y_min, p.position.y);
  contact_point.z = min(z_max, max(z_min, p.position.z));

  char maxComponent = 'y';
  float maxDepth    = y_min - p.position.y;

  if (maxDepth < x_min - p.position.x) {
      maxComponent = 'x';
      maxDepth = x_min - p.position.x;
  } else if (maxDepth < p.position.x - x_max) {
      maxComponent = 'x';
      maxDepth = p.position.x - x_max;
  }

  if (maxDepth < z_min - p.position.z) {
      maxComponent = 'z';
      maxDepth = z_min - p.position.z;
  } else if (maxDepth < p.position.z - z_max) {
      maxComponent = 'z';
      maxDepth = p.position.z - z_max;
  }

  float3 surface_normal = make_float3(0.0f);
  switch (maxComponent) {
    case 'x':
      if (p.position.x < x_min) {
          surface_normal = make_float3(1.0f,  0.0f,  0.0f);
      }
      else if (p.position.x > x_max) {
          surface_normal = make_float3(-1.0f,  0.0f,  0.0f);
      }
      break;
    case 'y':
      if (p.position.y < y_min) {
          surface_normal = make_float3(0.0f,  1.0f,  0.0f);
      }
      break;
    case 'z':
      if (p.position.z < z_min) {
          surface_normal = make_float3(0.0f,  0.0f,  1.0f);
      }
      else if (p.position.z > z_max) {
          surface_normal = make_float3(0.0f,  0.0f, -1.0f);
      }
      break;
  }

  // eq 4.58
  float penetration_depth = optix::length(p.position - contact_point);
  p.velocity = p.velocity - (1.0f + restitution * penetration_depth / (dt * optix::length(p.velocity))) * optix::dot(p.velocity, surface_normal) * surface_normal;
  p.position = contact_point;
}

// p54
RT_PROGRAM void update() {
    Particle& p = particles_buffer[launch_index];

    // Find nearest neighbors
    unsigned int nn_count = 0;
    unsigned int nn[3 * 3 * 3 * HASH_CELL_SIZE];
    nearest_neighbor_search(p, nn_count, nn);

    float3 tot_force = make_float3(0.0f);

    // Internal forces
    tot_force += pressure_force(p, nn_count, nn);
    tot_force += viscosity_force(p, nn_count, nn);

    // External forces
    tot_force += gravity_force(p.density);
    tot_force += surface_tension_force(p, nn_count, nn);

    // Integrate forces over time
    euler_cromer(p, tot_force);

    // Handle potential collisions
    collision_detection(p);
}