#include "hip/hip_runtime.h"
#include "common.cuh"

// The vertex buffer and index buffer of the geometry to test against.
rtBuffer<VertexData> vertex_buffer;
rtBuffer<uint3>      index_buffer;

// Builds an AABB for geometry made of indexed triangle data.
RT_PROGRAM void boundingbox_triangle_indexed(int primitive_index, float result[6]) {
  optix::Aabb *aabb = (optix::Aabb *) result;

  // Triangle vertices.
  const uint3 indices = index_buffer[primitive_index];
  const float3 v0 = vertex_buffer[indices.x].position;
  const float3 v1 = vertex_buffer[indices.y].position;
  const float3 v2 = vertex_buffer[indices.z].position;

  const float area = optix::length(optix::cross(v1 - v0, v2 - v0)) / 2.0f;

  if (area <= 0.0f || isinf(area)) {
    aabb->invalidate();
  }
  
  aabb->m_min = fminf(fminf(v0, v1), v2); // Combine the smallest component values into the min boundary point.
  aabb->m_max = fmaxf(fmaxf(v0, v1), v2); // Combine the greatest component values into the max boundary point.
}
