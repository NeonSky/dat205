#include "hip/hip_runtime.h"
#include "common.cuh"

// The 2D color (RGBA32F) buffer we will render our result to.
rtBuffer<float4, 2> output_buffer;

// Current pixel index.
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

RT_PROGRAM void exception() {

  // See common.cuh
  #if USE_DEBUG_EXCEPTIONS

  const unsigned int err_code = rtGetExceptionCode();
  if (RT_EXCEPTION_USER <= err_code) {
    rtPrintf("User exception %d at (%d, %d)\n", err_code - RT_EXCEPTION_USER, launch_index.x, launch_index.y);
  } else {
    rtPrintf("Exception code 0x%X at (%d, %d)\n", err_code, launch_index.x, launch_index.y);
  }

  // Write a very bright magenta as error color (makes sure that the error color is not accumulated away in a progressive renderer).
  output_buffer[launch_index] = make_float4(1000000.0f, 0.0f, 1000000.0f, 1.0f);

  #endif
}