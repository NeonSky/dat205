#include "hip/hip_runtime.h"
#include "common.cuh"

// The vertex buffer and index buffer of the geometry to test against.
rtBuffer<VertexData> vertex_buffer;
rtBuffer<uint3>      index_buffer;

// The current ray.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

// Output attributes that we will define/write upon an intersection.
rtDeclareVariable(optix::float3, attr_geo_normal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, attr_tangent,    attribute TANGENT   , );
rtDeclareVariable(optix::float3, attr_normal,     attribute NORMAL    , );
rtDeclareVariable(optix::float3, attr_uv,         attribute TEX_UV    , );

// Checks for intersection against geometry made of indexed triangle data.
RT_PROGRAM void intersection_triangle_indexed(int primitive_index) {

  // Triangle vertices.
  const uint3 indices = index_buffer[primitive_index];
  VertexData const& v0 = vertex_buffer[indices.x];
  VertexData const& v1 = vertex_buffer[indices.y];
  VertexData const& v2 = vertex_buffer[indices.z];

  // Ray-triangle intersection test.
  float3 n;
  float  t;
  float  beta;
  float  gamma;

  if (intersect_triangle(ray, v0.position, v1.position, v2.position, n, t, beta, gamma)) { // NOTE: intersect_triangle() is defined in optixu_math_namespace.h. See: https://docs.nvidia.com/gameworks/content/gameworkslibrary/optix/optixapireference/optixu__math__namespace_8h.html
    if (rtPotentialIntersection(t)) {

      // Barycentric interpolation
      const float alpha = 1.0f - beta - gamma;

      // NOTE: We will normalize the results in the hit shaders.
      attr_geo_normal = n;
      attr_tangent    = v0.tangent  * alpha + v1.tangent  * beta + v2.tangent  * gamma;
      attr_normal     = v0.normal   * alpha + v1.normal   * beta + v2.normal   * gamma;
      attr_uv         = v0.uv       * alpha + v1.uv       * beta + v2.uv       * gamma;
      
      // Report intersection for material 0.
      rtReportIntersection(0);
    }
  }
}
