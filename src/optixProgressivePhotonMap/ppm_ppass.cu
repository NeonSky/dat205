#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "ppm.h"
#include "random.h"

using namespace optix;

//
// Scene wide variables
//
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );


//
// Ray generation program
//
rtBuffer<PhotonRecord, 1>        ppass_output_buffer;
rtBuffer<uint2, 2>               photon_rnd_seeds;
rtDeclareVariable(uint,          max_depth, , );
rtDeclareVariable(uint,          max_photon_count, , );
rtDeclareVariable(PPMLight,      light , , );

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );


static __device__ __inline__ float2 rnd_from_uint2( uint2& prev )
{
  return make_float2(rnd(prev.x), rnd(prev.y));
}

static __device__ __inline__ void mapToDisk( optix::float2& sample )
{
  float phi, r;
  float a = 2.0f * sample.x - 1.0f;      // (a,b) is now on [-1,1]^2 
  float b = 2.0f * sample.y - 1.0f;      // 
  if (a > -b) {                           // reg 1 or 2 
    if (a > b) {                          // reg 1, also |a| > |b| 
      r = a;
      phi = (M_PIf/4.0f) * (b/a);
    } else {                              // reg 2, also |b| > |a| 
      r = b;
      phi = (M_PIf/4.0f) * (2.0f - a/b);
    }
  } else {                                // reg 3 or 4 
    if (a < b) {                          // reg 3, also |a|>=|b| && a!=0 
      r = -a;
      phi = (M_PIf/4.0f) * (4.0f + b/a);
    } else {                              // region 4, |b| >= |a|,  but 
      // a==0 and  b==0 could occur. 
      r = -b;
      phi = b != 0.0f ? (M_PIf/4.0f) * (6.0f - a/b) :
        0.0f;
    }
  }
  float u = r * cosf( phi );
  float v = r * sinf( phi );
  sample.x = u;
  sample.y = v;
}

// sample hemisphere with cosine density
static __device__ __inline__ void sampleUnitHemisphere( const optix::float2& sample,
                                                 const optix::float3& U,
                                                 const optix::float3& V,
                                                 const optix::float3& W,
                                                 optix::float3& point )
{
    float phi = 2.0f * M_PIf*sample.x;
    float r = (float)sqrt( sample.y );
    float x = r * (float)cos(phi);
    float y = r * (float)sin(phi);
    float z = 1.0f - x*x -y*y;
    z = z > 0.0f ? (float)sqrt(z) : 0.0f;

    point = x*U + y*V + z*W;
}

static __device__ __inline__ void generateAreaLightPhoton( const PPMLight& light, const float2& d_sample, float3& o, float3& d)
{
  // Choose a random position on light
  o = light.anchor + 0.5f * ( light.v1 + light.v2);
  
  // Choose a random direction from light
  float3 U, V, W;
  create_onb( light.direction, U, V, W);
  sampleUnitHemisphere( d_sample, U, V, W, d );
}

static __device__ __inline__ void generateSpotLightPhoton( const PPMLight& light, const float2& d_sample, float3& o, float3& d)
{
  o = light.position;

  // Choose random dir by sampling disk of radius light.radius and projecting up to unit hemisphere
  float2 square_sample = d_sample; 
  mapToDisk( square_sample );
  square_sample = square_sample * atanf( light.radius );
  float x = square_sample.x;
  float y = square_sample.y;
  float z = sqrtf( fmaxf( 0.0f, 1.0f - x*x - y*y ) );

  // Now transform into light space
  float3 U, V, W;
  create_onb(light.direction, U, V, W);
  d =  x*U + y*V + z*W;
}


RT_PROGRAM void ppass_camera()
{
  size_t2 size     = photon_rnd_seeds.size();
  uint    pm_index = (launch_index.y * size.x + launch_index.x) * max_photon_count;
  uint2   seed     = photon_rnd_seeds[launch_index]; // No need to reset since we dont reuse this seed

  float2 direction_sample = make_float2(
      ( static_cast<float>( launch_index.x ) + rnd( seed.x ) ) / static_cast<float>( size.x ),
      ( static_cast<float>( launch_index.y ) + rnd( seed.y ) ) / static_cast<float>( size.y ) );
  float3 ray_origin, ray_direction;
  if( light.is_area_light ) {
    generateAreaLightPhoton( light, direction_sample, ray_origin, ray_direction );
  } else {
    generateSpotLightPhoton( light, direction_sample, ray_origin, ray_direction );
  }

  optix::Ray ray(ray_origin, ray_direction, ppass_and_gather_ray_type, scene_epsilon );

  // Initialize our photons
  for(unsigned int i = 0; i < max_photon_count; ++i) {
    ppass_output_buffer[i+pm_index].energy = make_float3(0.0f);
  }

  PhotonPRD prd;
  //  rec.ray_dir = ray_direction; // set in ppass_closest_hit
  prd.energy = light.power;
  prd.sample = seed;
  prd.pm_index = pm_index;
  prd.num_deposits = 0;
  prd.ray_depth = 0;
  rtTrace( top_object, ray, prd );
}

//
// Closest hit material
//
rtDeclareVariable(float3,  Ks, , )={0,0,0};
rtDeclareVariable(float3,  Kd, , )={0.7,0.7,0.7};
rtDeclareVariable(float3,  emitted, , )={0,0,0};
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, ); 
rtDeclareVariable(float3, shading_normal, attribute shading_normal, ); 
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(PhotonPRD, hit_record, rtPayload, );

RT_PROGRAM void ppass_closest_hit()
{
  // Check if this is a light source
  float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
  float3 ffnormal     = faceforward( world_shading_normal, -ray.direction, world_geometric_normal );

  float3 hit_point = ray.origin + t_hit*ray.direction;
  float3 new_ray_dir;

  if( fmaxf( Kd ) > 0.0f ) {
    // We hit a diffuse surface; record hit if it has bounced at least once
    if( hit_record.ray_depth > 0 ) {
      PhotonRecord& rec = ppass_output_buffer[hit_record.pm_index + hit_record.num_deposits];
      rec.position = hit_point;
      rec.normal = ffnormal;
      rec.ray_dir = ray.direction;
      rec.energy = hit_record.energy;
      hit_record.num_deposits++;
    }

    hit_record.energy = Kd * hit_record.energy; 
    float3 U, V, W;
    create_onb(ffnormal, U, V, W);
    sampleUnitHemisphere(rnd_from_uint2(hit_record.sample), U, V, W, new_ray_dir);

  } else {
    hit_record.energy = Ks * hit_record.energy;
    // Make reflection ray
    new_ray_dir = reflect( ray.direction, ffnormal );
  }

  hit_record.ray_depth++;
  if ( hit_record.num_deposits >= max_photon_count || hit_record.ray_depth >= max_depth)
    return;

  optix::Ray new_ray( hit_point, new_ray_dir, ppass_and_gather_ray_type, scene_epsilon );
  rtTrace(top_object, new_ray, hit_record);
}

