#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013-2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include "app_config.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "rt_function.h"
#include "per_ray_data.h"
#include "light_definition.h"
#include "shader_common.h"

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

rtBuffer<LightDefinition> sysLightDefinitions;

rtDeclareVariable(float, sysEnvironmentRotation, , );


// Not actually a light. Never appears inside the sysLightDefinitions.
RT_PROGRAM void miss_environment_null()
{
  thePrd.radiance = make_float3(0.0f);

  thePrd.flags |= FLAG_TERMINATE;
}

RT_PROGRAM void miss_environment_constant()
{
#if USE_NEXT_EVENT_ESTIMATION
  // If the last surface intersection was a diffuse which was directly lit with multiple importance sampling,
  // then calculate light emission with multiple importance sampling as well.
  const float weightMIS = (thePrd.flags & FLAG_DIFFUSE) ? powerHeuristic(thePrd.pdf, 0.25f * M_1_PIf) : 1.0f;
  thePrd.radiance = make_float3(weightMIS); // Constant white emission multiplied by MIS weight.
#else
  thePrd.radiance = make_float3(1.0f); // Constant white emission.
#endif

  thePrd.flags |= FLAG_TERMINATE;
}

