#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

///////////////////////////////////////////////////////////////////////////////

#include "helpers.h"
#include <optix.h>
#include <optix_math.h>


rtDeclareVariable( uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable( uint2, launch_dim,   rtLaunchDim, );

rtDeclareVariable( float, f_exposure, , );

rtBuffer<float4, 2> pre_image;
rtBuffer<uchar4, 2> output_buffer;



RT_PROGRAM void tonemap()
{
  float3 val_Yxy = rgb2Yxy( make_float3( pre_image[ launch_index ] ) );
  
  float Y        = val_Yxy.x; // Y channel is luminance
  float mapped_Y = Y / ( Y + 1.0f );
  
  float3 mapped_Yxy = make_float3( mapped_Y, val_Yxy.y, val_Yxy.z ); 
  float3 mapped_rgb = Yxy2rgb( mapped_Yxy ); 

  output_buffer[ launch_index ] = make_color( mapped_rgb );  
}


