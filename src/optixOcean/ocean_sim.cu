#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optix_math.h>
#include <hipfft/hipfft.h>
#include <hip/hip_math_constants.h>


rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, patch_size,, );
rtDeclareVariable(float, t,, );
rtBuffer<float2, 2>                    h0;
rtBuffer<float2, 2>                    ht;
rtBuffer<float2, 2>                    ik_ht;


/******************************************************************************\
 * 
 * Frequency space spectrum generation 
 * 
\******************************************************************************/
// complex math functions
__device__
float2 conjugate(float2 arg)
{ return make_float2(arg.x, -arg.y); }

__device__
float2 complex_exp(float arg)
{ return make_float2(cosf(arg), sinf(arg)); }

__device__
float2 complex_add(float2 a, float2 b)
{ return make_float2(a.x + b.x, a.y + b.y); }

__device__
float2 complex_mult(float2 ab, float2 cd)
{ return make_float2(ab.x * cd.x - ab.y * cd.y, ab.x * cd.y + ab.y * cd.x); }

RT_PROGRAM void generate_spectrum()
{
    unsigned int x = launch_index.x; 
    unsigned int y = launch_index.y;
    
    // calculate coordinates
    float2 k;
    k.x = HIP_PI_F * x / patch_size;
    k.y = 2.0f * HIP_PI_F * y / patch_size;

    // calculate dispersion w(k)
    float k_len = sqrtf( k.x*k.x + k.y*k.y );
    float w = sqrtf( 9.81f * k_len );

    float2 h0_k  = h0[ make_uint2( x, y ) ];
    float2 h0_mk = h0[ make_uint2( x, launch_dim.y-1-y ) ];

    float2 h_tilda = complex_add( complex_mult(h0_k, complex_exp(w * t)),
                                  complex_mult(conjugate(h0_mk), complex_exp(-w * t)) );
    float2 ik_h_tilda = k*h_tilda;

    ht[ launch_index ] = h_tilda;
    ik_ht[ launch_index ] = ik_h_tilda;
}


/******************************************************************************\
 * 
 * Normal calculation 
 * 
\******************************************************************************/
rtBuffer<float,  2>                    heights;
rtBuffer<float4, 2>                    normals;

rtDeclareVariable(float, height_scale, , );

RT_PROGRAM void calculate_normals()
{
    unsigned int x = launch_index.x; 
    unsigned int y = launch_index.y;
    unsigned int width  = launch_dim.x;
    unsigned int height = launch_dim.y;

    float2 slope;
    if ( (x > 0u) && ( y > 0u ) && ( x < width-1u ) && ( y < height-1u ) ) {
      slope.x = heights[ make_uint2( x+1, y   ) ]- heights[ make_uint2( x-1, y   ) ];
      slope.y = heights[ make_uint2( x,   y+1 ) ]- heights[ make_uint2( x,   y-1 ) ];
    } else {
      slope = make_float2(0.0f, 0.0f);
    }
    float3 normal = normalize( cross( make_float3( 0.0f,          slope.y*height_scale, 2.0f / width ),
                                      make_float3( 2.0f / height, slope.x*height_scale, 0.0f         ) ) );
    normals[launch_index] = make_float4( normal, 0.0f );
}

