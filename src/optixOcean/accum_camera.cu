#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "random.h"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  int depth;
  unsigned int seed;
};

struct PerRayData_shadow
{
    float3 attenuation;
};


rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , );
rtBuffer<uchar4, 2>              output_buffer;
rtBuffer<float4, 2>              accum_buffer;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  frame, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );


RT_PROGRAM void pinhole_camera()
{

  size_t2 screen = output_buffer.size();
  unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame);

  // Subpixel jitter: send the ray through a different position inside the pixel each time,
  // to provide antialiasing.
  float2 subpixel_jitter = frame == 0 ? make_float2(0.0f, 0.0f) : make_float2(rnd( seed ) - 0.5f, rnd( seed ) - 0.5f);

  float2 d = (make_float2(launch_index) + subpixel_jitter) / make_float2(screen) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  
  optix::Ray ray(ray_origin, ray_direction, /*radiance ray type*/ 0, scene_epsilon );

  PerRayData_radiance prd;
  prd.depth = 0;
  prd.seed = seed;

  rtTrace(top_object, ray, prd);

  float4 acc_val = accum_buffer[launch_index];
  if( frame > 0 ) {
    acc_val = lerp( acc_val, make_float4( prd.result, 0.f), 1.0f / static_cast<float>( frame+1 ) );
  } else {
    acc_val = make_float4(prd.result, 0.f);
  }
  output_buffer[launch_index] = make_color( make_float3( acc_val ) );
  accum_buffer[launch_index] = acc_val;
}

RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d,%d)\n", code, launch_index.x, launch_index.y );
  output_buffer[launch_index] = make_color( bad_color );
  accum_buffer[launch_index] = make_float4( bad_color, 1.0f );
}

