#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<VertexData> vertex_buffer;
rtBuffer<uint3>      index_buffer;

// Axis Aligned Bounding Box routine for indexed interleaved triangle data.
RT_PROGRAM void boundingbox_triangle_indexed(int primitiveIndex, float result[6]) {
  const uint3 indices = index_buffer[primitiveIndex];

  const float3 v0 = vertex_buffer[indices.x].position;
  const float3 v1 = vertex_buffer[indices.y].position;
  const float3 v2 = vertex_buffer[indices.z].position;

  const float area = optix::length(optix::cross(v1 - v0, v2 - v0));

  optix::Aabb *aabb = (optix::Aabb *) result;

  if (0.0f < area && !isinf(area)) {
    aabb->m_min = fminf(fminf(v0, v1), v2);
    aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
  } else {
    aabb->invalidate();
  }
}
