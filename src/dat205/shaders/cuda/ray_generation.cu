#include "hip/hip_runtime.h"
#include "common.cuh"

// The 2D color (RGBA32F) buffer we will render our result to.
rtBuffer<float4,  2> output_buffer;

// Root object of the scene.
rtDeclareVariable(rtObject, root, , );

// Output buffer/screen dimensionality and current pixel index.
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim  , );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, ); // NOTE: launchIndex = (0, 0) is the bot-left corner (thus, matches OpenGL's texture origin) of the buffer/screen.

// Camera.
rtDeclareVariable(float3, camera_pos    , , );
rtDeclareVariable(float3, camera_right  , , );
rtDeclareVariable(float3, camera_up     , , );
rtDeclareVariable(float3, camera_forward, , );

// Entry point for this ray tracing kernel.
RT_PROGRAM void ray_generation() {
  
  const float2 pixel = make_float2(launch_index);
  const float2 pixel_center = pixel + make_float2(0.5f);

  const float2 screen = make_float2(launch_dim);
  const float2 ndc = (pixel_center / screen) * 2.0f - 1.0f; // [-1, 1] Normalized Device Coordinates

  // Pinhole camera model.
  const float3 origin    = camera_pos;
  const float3 direction = optix::normalize(ndc.x * camera_right + ndc.y * camera_up + camera_forward); // NOTE: the direction must be normalized.

  // Setup ray from origin, towards direction, using ray type 0, and test the interval between 0.0f and RT_DEFAULT_MAX for intersections.
  optix::Ray ray = optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);

  // Shoot the ray, storing the result in payload.
  RayPayload payload;
  payload.radiance        = make_float3(0.0f);
  payload.importance      = 1.0f;
  payload.recursion_depth = 0;

  rtTrace(root, ray, payload);

  // Write result to output buffer.
  output_buffer[launch_index] = make_float4(payload.radiance, 1.0f);
}