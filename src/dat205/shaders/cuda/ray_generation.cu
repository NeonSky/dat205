#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<float4,  2> output_buffer; // RGBA32F

rtDeclareVariable(rtObject, root, , );

rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtDeclareVariable(float3, camera_pos, , );
rtDeclareVariable(float3, camera_right, , );
rtDeclareVariable(float3, camera_up, , );
rtDeclareVariable(float3, camera_forward, , );

// Entry point for simple color filling kernel.
RT_PROGRAM void ray_generation() {

  RayPayload payload;

  payload.radiance = make_float3(0.0f);
  
  // The launch index is the pixel coordinate.
  // Note that launchIndex = (0, 0) is the bottom left corner of the image,
  // which matches the origin in the OpenGL texture used to display the result.
  const float2 pixel = make_float2(launch_index);
  // Sample the ray in the center of the pixel.
  const float2 fragment = pixel + make_float2(0.5f);
  // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
  const float2 screen = make_float2(launch_dim);
  // Normalized device coordinates in range [-1, 1].
  const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

  const float3 origin    = camera_pos;
  const float3 direction = optix::normalize(ndc.x * camera_right + ndc.y * camera_up + camera_forward);

  // Shoot a ray from origin into direction (must always be normalized!) for ray type 0 and test the interval between 0.0f and RT_DEFAULT_MAX for intersections.
  optix::Ray ray = optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);

  // Start the ray traversal at the scene's root node.
  // The ray becomes the variable with rtCurrentRay semantic in the other program domains.
  // The PerRayData becomes the variable with the semantic rtPayload in the other program domains,
  // which allows to exchange arbitrary data between the program domains.
  rtTrace(root, ray, payload);

  output_buffer[launch_index] = make_float4(payload.radiance, 1.0f);
}