#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "common.cuh"

rtBuffer<float4,  2> sysOutputBuffer; // RGBA32F

rtDeclareVariable(rtObject, sysTopObject, , );

rtDeclareVariable(uint2, theLaunchDim,   rtLaunchDim, );
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );

rtDeclareVariable(float3, sysCameraPosition, , );
rtDeclareVariable(float3, sysCameraU, , );
rtDeclareVariable(float3, sysCameraV, , );
rtDeclareVariable(float3, sysCameraW, , );

// Entry point for simple color filling kernel.
RT_PROGRAM void raygeneration() {

  PerRayData prd;

  prd.radiance = make_float3(0.0f);
  
  // The launch index is the pixel coordinate.
  // Note that launchIndex = (0, 0) is the bottom left corner of the image,
  // which matches the origin in the OpenGL texture used to display the result.
  const float2 pixel = make_float2(theLaunchIndex);
  // Sample the ray in the center of the pixel.
  const float2 fragment = pixel + make_float2(0.5f);
  // The launch dimension (set with rtContextLaunch) is the full client window in this demo's setup.
  const float2 screen = make_float2(theLaunchDim);
  // Normalized device coordinates in range [-1, 1].
  const float2 ndc = (fragment / screen) * 2.0f - 1.0f;

  const float3 origin    = sysCameraPosition;
  const float3 direction = optix::normalize(ndc.x * sysCameraU + ndc.y * sysCameraV + sysCameraW);

  // Shoot a ray from origin into direction (must always be normalized!) for ray type 0 and test the interval between 0.0f and RT_DEFAULT_MAX for intersections.
  optix::Ray ray = optix::make_Ray(origin, direction, 0, 0.0f, RT_DEFAULT_MAX);

  // Start the ray traversal at the scene's root node.
  // The ray becomes the variable with rtCurrentRay semantic in the other program domains.
  // The PerRayData becomes the variable with the semantic rtPayload in the other program domains,
  // which allows to exchange arbitrary data between the program domains.
  rtTrace(sysTopObject, ray, prd);

  sysOutputBuffer[theLaunchIndex] = make_float4(prd.radiance, 1.0f);
}