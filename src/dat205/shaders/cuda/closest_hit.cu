#include "hip/hip_runtime.h"
#include "common.cuh"

// Root object of the scene.
rtDeclareVariable(rtObject, root, , );

// The current ray and its payload.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay , );
rtDeclareVariable(RayPayload, payload, rtPayload, );

// The distance from the ray origin to where the intersection was detected.
rtDeclareVariable(float, ray_t, rtIntersectionDistance, );

// Point lights in the scene.
rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<PointLight> lights;

// Properties of the hit surface's material.
rtDeclareVariable(float3, mat_emissive_coefficient, , );
rtDeclareVariable(float3, mat_ambient_coefficient , , );
rtDeclareVariable(float3, mat_diffuse_coefficient , , );
rtDeclareVariable(float3, mat_specular_coefficient, , );
rtDeclareVariable(float, mat_refractive_index     , , );
rtDeclareVariable(float, mat_fresnel              , , );
rtDeclareVariable(float, mat_transparency         , , );

// Attributes from intersection test.
rtDeclareVariable(optix::float3, attr_geo_normal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, attr_tangent,    attribute TANGENT   , );
rtDeclareVariable(optix::float3, attr_normal,     attribute NORMAL    , );
rtDeclareVariable(optix::float3, attr_uv,         attribute TEX_UV    , );

RT_PROGRAM void closest_hit() {

  // Transform the (unnormalized) object space normals into world space.
  float3 geo_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_geo_normal));
  float3 normal     = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));

  // Base color, regardless if the surface is exposed to light or not.
  float3 color = mat_emissive_coefficient + mat_ambient_coefficient * ambient_light_color;

  // Flip the shading normal if we hit the backface of the triangle.
  if (optix::dot(-ray.direction, geo_normal) < 0.0f) {
    normal = -normal;
  }

  float3 hit = ray.origin + ray_t * ray.direction;

  // Direct illumination
  for (int i = 0; i < lights.size(); i++) {
    PointLight light = lights[i];
    float3 light_vec = optix::normalize(light.position - hit);

    // Add light from light if the lights is on the same side of the surface.
    float n_dot_l = optix::dot(normal, light_vec);
    if (0.0f < n_dot_l) {

      // Setup a shadow ray from the hit/intersection point, towards the current point light.
      float dist_to_light = optix::length(light.position - hit);
      optix::Ray shadow_ray(hit, light_vec, 1, SHADOW_EPSILON, dist_to_light);

      // Shoot the shadow ray
      ShadowRayPayload shadow_payload;
      shadow_payload.attenuation = make_float3(1.0f);

      rtTrace(root, shadow_ray, shadow_payload);

      if (0.0f < fmaxf(shadow_payload.attenuation)) {
        float3 light_color = shadow_payload.attenuation * light.color;
        color += mat_diffuse_coefficient * n_dot_l * light_color;

        // Phong highlight
        float3 halfway_vec = optix::normalize((-ray.direction) + light_vec);
        float n_dot_h = optix::dot(normal, halfway_vec);
        if (0 < n_dot_h) {
          float highlight_sharpness = 88.0f;
          color += mat_specular_coefficient * light.color * pow(n_dot_h, highlight_sharpness);
        }
      }

    }
  }

  // Indirect illumination from reflections
  if (0.0f < mat_fresnel) {
    float3 reflection_vec = optix::reflect(ray.direction, normal);
    float3 halfway_vec = optix::normalize((-ray.direction) + reflection_vec);

    // Fresnel
    float wo_dot_h = optix::dot(-ray.direction, halfway_vec);
    float F = mat_fresnel + (1.0f - mat_fresnel) * pow(1.0f - wo_dot_h, 5.0f);

    float importance = payload.importance * optix::luminance(make_float3(F));

    float importance_threshold = 0.01f;
    unsigned int max_depth = 5;
    if (importance_threshold <= importance && payload.recursion_depth < max_depth) {

      // Setup a reflection ray from the hit/intersection point
      optix::Ray reflection_ray(hit, reflection_vec, 0, EPSILON, RT_DEFAULT_MAX);

      // Shoot the reflection ray
      RayPayload reflection_payload;
      reflection_payload.importance      = importance;
      reflection_payload.recursion_depth = payload.recursion_depth + 1;

      rtTrace(root, reflection_ray, reflection_payload);

      color += F * reflection_payload.radiance;
    }
  }

  // Indirect illumination from refractions
  // TODO: randomize between reflection and refractions by sampling the Fresnel term. This will prevent ray branching.
  if (0.0f < mat_transparency) {
    float3 n = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));

    float3 refraction_vec;
    bool total_internal_reflection = !optix::refract(refraction_vec, ray.direction, n, mat_refractive_index);

    if (!total_internal_reflection) {

      // External or internal reflection?
      float cos_theta = optix::dot(ray.direction, n);
      if (cos_theta < 0.0f) {
        cos_theta = -cos_theta;
      } else {
        cos_theta = optix::dot(refraction_vec, n);
      }

      float F = mat_fresnel + (1.0f - mat_fresnel) * pow(1.0f - cos_theta, 5.0f);

      float importance = payload.importance * (1.0f - F) * optix::luminance(make_float3(1.0f));

      const float importance_threshold = 0.01f;
      const unsigned int max_depth = 5;
      if (importance_threshold <= importance && payload.recursion_depth < max_depth) {
        optix::Ray refraction_ray(hit, refraction_vec, 0, EPSILON, RT_DEFAULT_MAX);

        // Shoot the refraction ray
        RayPayload refraction_payload;
        refraction_payload.importance      = payload.importance;
        refraction_payload.recursion_depth = payload.recursion_depth + 1;

        rtTrace(root, refraction_ray, refraction_payload);

        color += mat_transparency * (1.0f - F) * refraction_payload.radiance;
      }
    }
  }

  payload.radiance = color;
}
