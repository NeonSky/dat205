#include "hip/hip_runtime.h"
#include "common.cuh"

// Root object of the scene.
rtDeclareVariable(rtObject, root, , );

// The current ray and its payload.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay , );
rtDeclareVariable(RayPayload, payload, rtPayload, );

// The distance from the ray origin to where the intersection was detected.
rtDeclareVariable(float, ray_t, rtIntersectionDistance, );

// Point lights in the scene.
rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<PointLight> lights;

// Properties of the hit surface's material.
rtDeclareVariable(float3, mat_ambient_coefficient , , );
rtDeclareVariable(float3, mat_diffuse_coefficient , , );
rtDeclareVariable(float3, mat_specular_coefficient, , );
rtDeclareVariable(float, mat_fresnel              , , );

// Attributes from intersection test.
rtDeclareVariable(optix::float3, attr_geo_normal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, attr_tangent,    attribute TANGENT   , );
rtDeclareVariable(optix::float3, attr_normal,     attribute NORMAL    , );
rtDeclareVariable(optix::float3, attr_uv,         attribute TEX_UV    , );

RT_PROGRAM void closest_hit() {

  // Transform the (unnormalized) object space normals into world space.
  float3 geo_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_geo_normal));
  float3 normal     = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));

  // Base color, regardless if the surface is exposed to light or not.
  float3 color = mat_ambient_coefficient * ambient_light_color;

  // Flip the shading normal if we hit the backface of the triangle.
  if (0.0f < optix::dot(ray.direction, geo_normal)) {
    normal = -normal;
  }

  float3 hit = ray.origin + ray_t * ray.direction;

  // Direct illumination
  for (int i = 0; i < lights.size(); i++) {
    PointLight light = lights[i];
    float3 light_vec = optix::normalize(light.position - hit);

    // Add light from light if the lights is on the same side of the surface.
    float n_dot_l = optix::dot(normal, light_vec);
    if (0.0f < n_dot_l) {

      // Setup a shadow ray from the hit/intersection point, towards the current point light.
      float dist_to_light = optix::length(light.position - hit);
      optix::Ray shadow_ray(hit, light_vec, 1, SHADOW_EPSILON, dist_to_light);

      // Shoot the shadow ray
      ShadowRayPayload shadow_payload;
      shadow_payload.attenuation = make_float3(1.0f);

      rtTrace(root, shadow_ray, shadow_payload);

      if (0.0f < fmaxf(shadow_payload.attenuation)) {
        float3 light_color = shadow_payload.attenuation * light.color;
        color += mat_diffuse_coefficient * n_dot_l * light_color;

        // Phong highlight
        float3 halfway_vec = optix::normalize((-ray.direction) + light_vec);
        float n_dot_h = optix::dot(normal, halfway_vec);
        if (0 < n_dot_h) {
          float highlight_sharpness = 88.0f;
          color += mat_specular_coefficient * light.color * pow(n_dot_h, highlight_sharpness);
        }
      }

    }
  }

  // Indirect illumination
  if (0.0f < mat_fresnel) {

    float3 reflection_vec = optix::reflect(ray.direction, normal);
    float3 halfway_vec = optix::normalize((-ray.direction) + reflection_vec);

    // Fresnel
    float wi_dot_n = optix::dot(-ray.direction, halfway_vec);
    float F = mat_fresnel + (1.0f - mat_fresnel) * pow(1.0f - wi_dot_n, 5.0f);

    float importance = payload.importance * optix::luminance(make_float3(F));

    float importance_threshold = 0.01f;
    unsigned int max_depth = 5;
    if (importance_threshold <= importance && payload.recursion_depth < max_depth) {

      // Setup a reflection ray from the hit/intersection point
      optix::Ray reflection_ray(hit, reflection_vec, 0, EPSILON, RT_DEFAULT_MAX);

      // Shoot the reflection ray
      RayPayload reflection_payload;
      reflection_payload.importance      = importance;
      reflection_payload.recursion_depth = payload.recursion_depth + 1;

      rtTrace(root, reflection_ray, reflection_payload);

      color += F * reflection_payload.radiance;
    }
  }

  payload.radiance = color;
}
