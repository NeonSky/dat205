#include "hip/hip_runtime.h"
#include "common.cuh"

rtDeclareVariable(rtObject, root, , );

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(RayPayload, payload, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, attr_geo_normal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, attr_tangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, attr_normal,    attribute NORMAL, );
//rtDeclareVariable(optix::float3, attr_uv,  attribute TEXCOORD, ); 

// This closest hit program only uses the geometric normal and the shading normal attributes.
// OptiX will remove all code from the intersection programs for unused attributes automatically.

// Note that the matching between attribute outputs from the intersection program and 
// the inputs in the closesthit and anyhit programs is done with the type (here float3) and
// the user defined attribute semantic (e.g. here NORMAL). 
// The actual variable name doesn't need to match but it's recommended for clarity.

RT_PROGRAM void closest_hit() {

  // Transform the (unnormalized) object space normals into world space.
  float3 geo_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_geo_normal));
  float3 normal    = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));

  // Check if the ray hit the geometry on the frontface or the backface.
  // The geometric normal is always defined on the front face of the geometry.
  // In this implementation the coordinate systems are right-handed and the frontface triangle winding is counter-clockwise (matching OpenGL).

  // If ray.direction and geometric normal are in the same hemisphere we're looking at a backface.
  if (0.0f < optix::dot(ray.direction, geo_normal)) {
    // Flip the shading normal to the backface, because only that is used below.
    // (See later examples for more intricate handling of the frontface condition.)
    normal = -normal;
  }

  // Visualize the resulting world space normal on the surface we're looking on.
  // Transform the normal components from [-1.0f, 1.0f] to the range [0.0f, 1.0f] to get colors for negative values.
  payload.radiance = normal * 0.5f + 0.5f;
}
