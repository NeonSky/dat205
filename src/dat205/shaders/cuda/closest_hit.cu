#include "hip/hip_runtime.h"
#include "common.cuh"

// Root object of the scene.
rtDeclareVariable(rtObject, root, , );

// The current ray and its payload.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay , );
rtDeclareVariable(RayPayload, payload, rtPayload, );

// The distance from the ray origin to where the intersection was detected.
rtDeclareVariable(float, ray_t, rtIntersectionDistance, );

// Point lights in the scene.
rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<PointLight> lights;

// Properties of the hit surface's material.
rtDeclareVariable(float3, mat_ambient_coefficient, , );
rtDeclareVariable(float3, mat_diffuse_coefficient, , );
rtDeclareVariable(float3, mat_specular_coefficient, , );

// Attributes from intersection test.
rtDeclareVariable(optix::float3, attr_geo_normal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, attr_tangent,    attribute TANGENT   , );
rtDeclareVariable(optix::float3, attr_normal,     attribute NORMAL    , );
rtDeclareVariable(optix::float3, attr_uv,         attribute TEX_UV    , );

RT_PROGRAM void closest_hit() {

  // Transform the (unnormalized) object space normals into world space.
  float3 geo_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_geo_normal));
  float3 normal     = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));

  // Base color, regardless if the surface is exposed to light or not.
  float3 color = mat_ambient_coefficient * ambient_light_color;

  // Flip the shading normal if we hit the backface of the triangle.
  if (0.0f < optix::dot(ray.direction, geo_normal)) {
    normal = -normal;
  }

  float3 hit = ray.origin + ray_t * ray.direction;

  for (int i = 0; i < lights.size(); i++) {
    PointLight light = lights[i];
    float3 light_vec = optix::normalize(light.position - hit);

    // Add light from light if the lights is on the same side of the surface.
    float n_dot_l = optix::dot(normal, light_vec);
    if (0 < n_dot_l) {

      // Setup a shadow ray from the hit/intersection point, towards the current point light.
      float dist_to_light = optix::length(light.position - hit);
      optix::Ray shadow_ray(hit, light_vec, 1, EPSILON, dist_to_light);

      // Shoot the shadow ray
      RayPayload shadow_payload;
      shadow_payload.radiance = make_float3(1.0f);
      rtTrace(root, shadow_ray, shadow_payload);

      if (0.0f < fmaxf(shadow_payload.radiance)) {
        float3 light_color = shadow_payload.radiance * light.color;
        color += mat_diffuse_coefficient * n_dot_l * light_color;

        // Phong highlight
        float3 halfway_vec = optix::normalize((-ray.direction) + light_vec);
        float n_dot_h = optix::dot(normal, halfway_vec);
        if (0 < n_dot_h) {
          float highlight_sharpness = 88.0f;
          color += mat_specular_coefficient * light.color * pow(n_dot_h, highlight_sharpness);
        }
      }

    }
  }

  payload.radiance = color;
}
