#include "hip/hip_runtime.h"
#include "common.cuh"

// Root object of the scene.
rtDeclareVariable(rtObject, root, , );

// The current ray and its payload.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay , );
rtDeclareVariable(RayPayload, payload, rtPayload, );

// The distance from the ray origin to where the intersection was detected.
rtDeclareVariable(float, ray_t, rtIntersectionDistance, );

// Point lights in the scene.
rtDeclareVariable(float3, ambient_light_color, , );
rtBuffer<PointLight> lights;

// Properties of the hit surface's material.
rtDeclareVariable(float3, mat_ambient_coefficient, , );
rtDeclareVariable(float3, mat_diffuse_coefficient, , );

// Attributes from intersection test.
rtDeclareVariable(optix::float3, attr_geo_normal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, attr_tangent,    attribute TANGENT   , );
rtDeclareVariable(optix::float3, attr_normal,     attribute NORMAL    , );
rtDeclareVariable(optix::float3, attr_uv,         attribute TEX_UV    , );

RT_PROGRAM void closest_hit() {

  // Transform the (unnormalized) object space normals into world space.
  float3 geo_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_geo_normal));
  float3 normal     = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));

  // Base color, regardless if the surface is exposed to light or not.
  float3 color = mat_ambient_coefficient * ambient_light_color;

  // Flip the shading normal if we hit the backface of the triangle.
  if (0.0f < optix::dot(ray.direction, geo_normal)) {
    normal = -normal;
  }

  float3 hit = ray.origin + ray_t * ray.direction;

  for (int i = 0; i < lights.size(); i++) {
    PointLight light = lights[i];
    float3 light_vec = optix::normalize(light.position - hit);

    // Add light from light if the lights is on the same side of the surface.
    float n_dot_l = optix::dot(normal, light_vec);
    if (0 < n_dot_l) {
      color += mat_diffuse_coefficient * n_dot_l * light.color;
    }
  }

  payload.radiance = color;
}
