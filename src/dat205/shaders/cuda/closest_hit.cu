#include "hip/hip_runtime.h"
#include "common.cuh"

// Root object of the scene.
rtDeclareVariable(rtObject, root, , );

// The current ray and its payload.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay , );
rtDeclareVariable(RayPayload, payload, rtPayload, );

// The distance from the ray origin to where the intersection was detected.
rtDeclareVariable(float, ray_t, rtIntersectionDistance, );

// Point lights in the scene.
rtBuffer<PointLight> lights;

// Properties of the hit surface's material.
rtDeclareVariable(float3, mat_color          , , );
rtDeclareVariable(float, mat_emission        , , ); // 0 = no emission       , 1 = full emission
rtDeclareVariable(float, mat_metalness       , , ); // 0 = dieletric         , 1 = metal
rtDeclareVariable(float, mat_shininess       , , ); // 0 = smeared highlight , 1 = dense highlight
rtDeclareVariable(float, mat_transparency    , , ); // 0 = opaque            , 1 = transparent
rtDeclareVariable(float, mat_reflectivity    , , ); // 0 = diffuse           , 1 = mirror
rtDeclareVariable(float, mat_fresnel         , , ); // 0 = only absorptions  , 1 = only reflections (when looking from directly above)
rtDeclareVariable(float, mat_refractive_index, , ); // 1.0 = air material

// Attributes from intersection test.
rtDeclareVariable(optix::float3, attr_geo_normal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, attr_tangent,    attribute TANGENT   , );
rtDeclareVariable(optix::float3, attr_normal,     attribute NORMAL    , );
rtDeclareVariable(optix::float3, attr_uv,         attribute TEX_UV    , );

RT_FUNCTION float fresnel(float wo_dot_h) {
  float F = mat_fresnel + (1.0f - mat_fresnel) * pow(1.0f - wo_dot_h, 5.0f); // Schlick's approximation.
  return F;
}

RT_FUNCTION float torrance_sparrow_brdf(float n_dot_wi,
                                        float n_dot_wo,
                                        float n_dot_wh,
                                        float wo_dot_wh) {

  float F = fresnel(wo_dot_wh);
  float D = ((mat_shininess + 2.0f) / (2.0f * M_PIf)) * pow(n_dot_wh, mat_shininess);
  float G = min(1.0f, min(2.0f * n_dot_wh * n_dot_wo / wo_dot_wh, 2.0f * n_dot_wh * n_dot_wi / wo_dot_wh));

  float denominator = 4.0f * n_dot_wo * n_dot_wi;

  return F * D * G / denominator;
}

RT_FUNCTION float3 direct_illumination(float3 const& wo,
                                       float3 const& hit,
                                       float3 const& n) {

  float3 illumination = make_float3(0.0f);

  // Do not illuminate the backface of a triangle.
  float n_dot_wo  = optix::dot(n, wo);
  if (n_dot_wo <= 0.0f) {
    return illumination;
  }

  for (int i = 0; i < lights.size(); i++) {
    PointLight light = lights[i];

    // Ensure that the light could illuminate the front face.
    float3 wi = optix::normalize(light.position - hit);
    float n_dot_wi = optix::dot(n, wi);
    if (n_dot_wi <= 0.0f) {
      continue;
    }

    // Setup a shadow ray from the hit/intersection point, towards the current point light.
    float dist_to_light = optix::length(light.position - hit);
    optix::Ray shadow_ray(hit, wi, 1, EPSILON, dist_to_light);

    // Shoot the shadow ray
    ShadowRayPayload shadow_payload;
    shadow_payload.attenuation = make_float3(1.0f);

    rtTrace(root, shadow_ray, shadow_payload);

    // Ensure that the shadow ray is not entirely absorbed along the way.
    if (fmaxf(shadow_payload.attenuation) <= 0.0f) {
      continue;
    }

    // Compute direct illumination from the light
    float3 light_illumination = shadow_payload.attenuation * (light.intensity / pow(dist_to_light, 2.0f)) * light.color;

    // Compute BRDF
    float3 wh = optix::normalize(wo + wi);

    float n_dot_wh  = optix::dot(n, wh);
    float wo_dot_wh = optix::dot(wo, wh);

    float F = fresnel(wo_dot_wh);
    float D = ((mat_shininess + 2.0f) / (2.0f * M_PIf)) * pow(n_dot_wh, mat_shininess);
    float G = min(1.0f, min(2.0f * n_dot_wh * n_dot_wo / wo_dot_wh, 2.0f * n_dot_wh * n_dot_wi / wo_dot_wh));
    float denominator = 4.0f * n_dot_wo * n_dot_wi;

    float brdf = F * D * G / denominator;

    // Material models
    float3 diffuse_model    = mat_color * M_1_PIf * n_dot_wi * light_illumination;
    float3 dieletric_model  = brdf * n_dot_wi * light_illumination + (1.0f - F) * diffuse_model;
    float3 metal_model      = brdf * mat_color * n_dot_wi * light_illumination;
    float3 microfacet_model = mat_metalness * metal_model + (1.0f - mat_metalness) * dieletric_model;

    // Apply a linear blend between a perfectly diffuse surface and a microfacet brdf.
    float3 material_model = mat_reflectivity * microfacet_model + (1.0f - mat_reflectivity) * diffuse_model;

    illumination += material_model;
  }

  return illumination;
}

RT_FUNCTION float3 indirect_illumination(float3 const& wo,
                                         float3 const& hit,
                                         float3 const& n) {

  float3 illumination = make_float3(0.0f);

  // Indirect illumination from reflections
  if (0.0f < mat_reflectivity && payload.recursion_depth < 3) {
    float3 wi = optix::reflect(ray.direction, n);
    float3 wh = optix::normalize(wo + wi);

    // Fresnel
    float wo_dot_wh = max(0.01f, optix::dot(wo, wh));
    float F = mat_reflectivity * fresnel(wo_dot_wh);

    float importance = payload.importance;
    float importance_threshold = 0.1f;
    if (importance_threshold <= importance) {

      // Setup a reflection ray from the hit/intersection point
      optix::Ray reflection_ray(hit, wi, 0, EPSILON, RT_DEFAULT_MAX);

      // Shoot the reflection ray
      RayPayload reflection_payload;
      reflection_payload.importance      = importance;
      reflection_payload.recursion_depth = payload.recursion_depth + 1;

      rtTrace(root, reflection_ray, reflection_payload);

      float3 mirror_model = F * reflection_payload.radiance;
      float3 metal_model  = mat_color * F * reflection_payload.radiance;

      float3 material_model = mat_metalness * metal_model + (1.0f - mat_metalness) * mirror_model;

      illumination += material_model;
    }
  }

  // Indirect illumination from refractions
  int max_recursion_depth = 5;
  if (0.0f < mat_transparency && payload.recursion_depth < max_recursion_depth) {
    float3 wi;
    bool total_internal_reflection = !optix::refract(wi, ray.direction, n, mat_refractive_index); // For optix::refract(), see https://docs.nvidia.com/gameworks/content/gameworkslibrary/optix/optixapireference/optixu__math__namespace_8h_source.html

    // float cos_theta = 0.0f;

    // Refraction with Schlick’s approximation reference (page 4): https://graphics.stanford.edu/courses/cs148-10-summer/docs/2006--degreve--reflection_refraction.pdf

    float F = 1.0f; // Fresnel of TIR

    if (total_internal_reflection) {
      wi = optix::reflect(ray.direction, n); // optix::refract() sets `wi = make_float3(0.0f)` on TIR, so we need to manually define `wi` in this case.
    }
    else {
      // External -> Internal (i.e. n1 <= n2)
      if(optix::dot(wo, n) <= 0.0f) {
        F = fresnel(optix::dot(wi, n));
      }

      // Internal -> External (i.e. n1 > n2)
      else {
        F = fresnel(optix::dot(wo, n));
      }
    }

    float importance = payload.importance * (1.0f - F);

    const float importance_threshold = 0.1f;
    if (importance_threshold <= importance) {
      optix::Ray refraction_ray(hit, wi, 0, EPSILON, RT_DEFAULT_MAX);

      // Shoot the refraction ray
      RayPayload refraction_payload;
      refraction_payload.importance      = payload.importance;
      refraction_payload.recursion_depth = payload.recursion_depth + 1;

      rtTrace(root, refraction_ray, refraction_payload);

      illumination += mat_transparency * (1.0f - F) * refraction_payload.radiance;
    }
  }

  return illumination;
}

RT_PROGRAM void closest_hit() {

  // Transform the (unnormalized) object space normals into world space.
  float3 geo_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_geo_normal));
  float3 normal     = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));

  float3 wo = -ray.direction;
  float3 hit = ray.origin + ray_t * ray.direction;

  // Base color, regardless if the surface is exposed to light or not.
  float3 color = make_float3(0.0f);

  color += mat_color * mat_emission;
  color += direct_illumination(wo, hit, normal);
  color += indirect_illumination(wo, hit, normal);

  payload.radiance = color;
}