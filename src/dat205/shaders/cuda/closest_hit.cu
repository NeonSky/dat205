#include "hip/hip_runtime.h"
#include "common.cuh"

// Root object of the scene.
rtDeclareVariable(rtObject, root, , );

// The current ray and its payload.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay , );
rtDeclareVariable(RayPayload, payload, rtPayload, );

// Attributes from intersection test.
rtDeclareVariable(optix::float3, attr_geo_normal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, attr_tangent,    attribute TANGENT   , );
rtDeclareVariable(optix::float3, attr_normal,     attribute NORMAL    , );
rtDeclareVariable(optix::float3, attr_uv,         attribute TEX_UV    , );

RT_PROGRAM void closest_hit() {

  // Transform the (unnormalized) object space normals into world space.
  float3 geo_normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_geo_normal));
  float3 normal     = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));

  // Flip the shading normal if we hit the backface of the triangle.
  if (0.0f < optix::dot(ray.direction, geo_normal)) {
    normal = -normal;
  }

  // Transform the normal components from [-1.0f, 1.0f] to the range [0.0f, 1.0f] and visualize as radiance.
  payload.radiance = normal * 0.5f + 0.5f;
}
