#include "hip/hip_runtime.h"
#include "common.cuh"

// The current ray payload.
rtDeclareVariable(ShadowRayPayload, payload, rtPayload, );

RT_PROGRAM void any_hit() {
  payload.attenuation = make_float3(0.0f);
  rtTerminateRay();
}
