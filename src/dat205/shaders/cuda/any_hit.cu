#include "hip/hip_runtime.h"
#include "common.cuh"

// The current ray payload.
rtDeclareVariable(RayPayload, payload, rtPayload, );

RT_PROGRAM void any_hit() {
  payload.radiance = make_float3(0.0f);
  rtTerminateRay();
}
