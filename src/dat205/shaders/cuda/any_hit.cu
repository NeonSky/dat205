#include "hip/hip_runtime.h"
#include "common.cuh"

// The current ray and its payload.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay , );
rtDeclareVariable(ShadowRayPayload, payload, rtPayload, );

// Properties of the hit surface's material.
rtDeclareVariable(float, mat_transparency, , );

// Attributes from intersection test.
rtDeclareVariable(optix::float3, attr_normal, attribute NORMAL, );

RT_PROGRAM void any_hit() {
  if (mat_transparency <= 0.0f) {
    payload.attenuation = make_float3(0.0f);
    rtTerminateRay();
  }

  // Reflect/absorb based on fresnel.
  float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));
  float n_dot_i = fabs(optix::dot(normal, ray.direction));

  float attenuation = 1.0f - mat_transparency;
  float F = attenuation + (1.0f - attenuation) * pow(1.0f - n_dot_i, 5.0f);

  payload.attenuation *= 1.0f - F;

  rtIgnoreIntersection();
}
