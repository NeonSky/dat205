#include "hip/hip_runtime.h"
#include "common.cuh"

// The current ray and its payload.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay , );
rtDeclareVariable(ShadowRayPayload, payload, rtPayload, );

// Properties of the hit surface's material.
rtDeclareVariable(float, mat_transparency, , );

// Attributes from intersection test.
rtDeclareVariable(optix::float3, attr_normal, attribute NORMAL, );

RT_PROGRAM void any_hit() {
  float3 normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, attr_normal));
  float n_dot_r = fabs(optix::dot(normal, ray.direction));

  float attenuation = 1.0f - mat_transparency;
  float F = attenuation + (1.0f - attenuation) * pow(1.0f - n_dot_r, 5.0f);

  payload.attenuation *= 1.0f - F;

  rtIgnoreIntersection();
}
