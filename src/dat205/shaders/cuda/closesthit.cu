#include "hip/hip_runtime.h"
#include "common.cuh"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
//rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, ); 

// This closest hit program only uses the geometric normal and the shading normal attributes.
// OptiX will remove all code from the intersection programs for unused attributes automatically.

// Note that the matching between attribute outputs from the intersection program and 
// the inputs in the closesthit and anyhit programs is done with the type (here float3) and
// the user defined attribute semantic (e.g. here NORMAL). 
// The actual variable name doesn't need to match but it's recommended for clarity.

RT_PROGRAM void closesthit()
{
  // Transform the (unnormalized) object space normals into world space.
  float3 geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
  float3 normal    = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));

  // Check if the ray hit the geometry on the frontface or the backface.
  // The geometric normal is always defined on the front face of the geometry.
  // In this implementation the coordinate systems are right-handed and the frontface triangle winding is counter-clockwise (matching OpenGL).

  // If theRay.direction and geometric normal are in the same hemisphere we're looking at a backface.
  if (0.0f < optix::dot(theRay.direction, geoNormal))
  {
    // Flip the shading normal to the backface, because only that is used below.
    // (See later examples for more intricate handling of the frontface condition.)
    normal = -normal;
  }

  // Visualize the resulting world space normal on the surface we're looking on.
  // Transform the normal components from [-1.0f, 1.0f] to the range [0.0f, 1.0f] to get colors for negative values.
  thePrd.radiance = normal * 0.5f + 0.5f;
}
