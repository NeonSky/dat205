#include "hip/hip_runtime.h"
#include "common.cuh"

// The current ray and its payload.
rtDeclareVariable(optix::Ray, ray, rtCurrentRay , );
rtDeclareVariable(RayPayload, payload, rtPayload, );

// Environment map texture sampler.
rtTextureSampler<float4, 2> env_map;

RT_PROGRAM void miss_environment_constant() {

  // Azimuth; angle from the ray's z-axis (ccw) to (x, z). Reference: https://www.wikiwand.com/en/Spherical_coordinate_system
  float theta = atan2f(ray.direction.x, ray.direction.z);

  // Altitude; angle from y-axis (down) to (x, y, z).
  float phi = 0.5f * M_PIf - acosf(ray.direction.y); // NOTE: no division by ray length, since it is normalized.

  // Derive texture coordinates. NOTE: M_1_PIf = 1 / M_PIf
  float u = 0.5f * theta * M_1_PIf;   // "theta" is in [0.0, 2pi], hence "0.5 * theta / pi" is in [0.0, 1.0]
  float v = 0.5f * (1.0f + sin(phi)); // "sin(phi)" is in [-1.0, 1.0], hence "0.5 + 0.5 * sin(phi)" is in [0.0, 1.0]

  // Artificially brighten the environment map a bit.
  float3 ambient_term = make_float3(0.1f, 0.1f, 0.1f);

  payload.radiance = make_float3(tex2D(env_map, u, v)) + ambient_term;
}
