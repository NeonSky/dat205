#include "hip/hip_runtime.h"
#include "common.cuh"

// The payload/context of the current ray.
rtDeclareVariable(RayPayload, payload, rtPayload, );

RT_PROGRAM void miss_environment_constant() {
  payload.radiance = make_float3(0.0f, 0.0f, 0.0f);
}
