#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<VertexData> vertexBuffer;
rtBuffer<uint3>      indicesBuffer;

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, ); 
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, ); 

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

// Intersection routine for indexed interleaved triangle data.
RT_PROGRAM void intersection_triangle_indexed(int primitiveIndex) {
  const uint3 indices = indicesBuffer[primitiveIndex];

  VertexData const& a0 = vertexBuffer[indices.x];
  VertexData const& a1 = vertexBuffer[indices.y];
  VertexData const& a2 = vertexBuffer[indices.z];

  const float3 v0 = a0.position;
  const float3 v1 = a1.position;
  const float3 v2 = a2.position;

  float3 n;
  float  t;
  float  beta;
  float  gamma;

  if (intersect_triangle(theRay, v0, v1, v2, n, t, beta, gamma))
  {
    if (rtPotentialIntersection(t))
    {
      // Barycentric interpolation:
      const float alpha = 1.0f - beta - gamma;

      // Note: No normalization on the TBN attributes here for performance reasons.
      //       It's done after the transformation into world space anyway.
      varGeoNormal      = n;
      varTangent        = a0.tangent  * alpha + a1.tangent  * beta + a2.tangent  * gamma;
      varNormal         = a0.normal   * alpha + a1.normal   * beta + a2.normal   * gamma;
      varTexCoord       = a0.texcoord * alpha + a1.texcoord * beta + a2.texcoord * gamma;
      
      rtReportIntersection(0);
    }
  }
}
