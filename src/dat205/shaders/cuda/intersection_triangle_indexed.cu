#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<VertexData> vertex_buffer;
rtBuffer<uint3>      index_buffer;

rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

rtDeclareVariable(optix::float3, attr_geo_normal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, attr_tangent,    attribute TANGENT, );
rtDeclareVariable(optix::float3, attr_normal,     attribute NORMAL, ); 
rtDeclareVariable(optix::float3, attr_uv,         attribute TEXCOORD, ); 

// Intersection routine for indexed interleaved triangle data.
RT_PROGRAM void intersection_triangle_indexed(int primitiveIndex) {
  const uint3 indices = index_buffer[primitiveIndex];

  VertexData const& a0 = vertex_buffer[indices.x];
  VertexData const& a1 = vertex_buffer[indices.y];
  VertexData const& a2 = vertex_buffer[indices.z];

  const float3 v0 = a0.position;
  const float3 v1 = a1.position;
  const float3 v2 = a2.position;

  float3 n;
  float  t;
  float  beta;
  float  gamma;

  if (intersect_triangle(ray, v0, v1, v2, n, t, beta, gamma)) {
    if (rtPotentialIntersection(t)) {
      // Barycentric interpolation:
      const float alpha = 1.0f - beta - gamma;

      // Note: No normalization on the TBN attributes here for performance reasons.
      //       It's done after the transformation into world space anyway.
      attr_geo_normal = n;
      attr_tangent    = a0.tangent  * alpha + a1.tangent  * beta + a2.tangent  * gamma;
      attr_normal     = a0.normal   * alpha + a1.normal   * beta + a2.normal   * gamma;
      attr_uv         = a0.uv       * alpha + a1.uv       * beta + a2.uv       * gamma;
      
      rtReportIntersection(0);
    }
  }
}
