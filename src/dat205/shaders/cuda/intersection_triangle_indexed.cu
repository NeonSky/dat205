#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<VertexAttributes> attributesBuffer;
rtBuffer<uint3>            indicesBuffer;

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, ); 
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, ); 

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

// Intersection routine for indexed interleaved triangle data.
RT_PROGRAM void intersection_triangle_indexed(int primitiveIndex) {
  const uint3 indices = indicesBuffer[primitiveIndex];

  VertexAttributes const& a0 = attributesBuffer[indices.x];
  VertexAttributes const& a1 = attributesBuffer[indices.y];
  VertexAttributes const& a2 = attributesBuffer[indices.z];

  const float3 v0 = a0.vertex;
  const float3 v1 = a1.vertex;
  const float3 v2 = a2.vertex;

  float3 n;
  float  t;
  float  beta;
  float  gamma;

  if (intersect_triangle(theRay, v0, v1, v2, n, t, beta, gamma))
  {
    if (rtPotentialIntersection(t))
    {
      // Barycentric interpolation:
      const float alpha = 1.0f - beta - gamma;

      // Note: No normalization on the TBN attributes here for performance reasons.
      //       It's done after the transformation into world space anyway.
      varGeoNormal      = n;
      varTangent        = a0.tangent  * alpha + a1.tangent  * beta + a2.tangent  * gamma;
      varNormal         = a0.normal   * alpha + a1.normal   * beta + a2.normal   * gamma;
      varTexCoord       = a0.texcoord * alpha + a1.texcoord * beta + a2.texcoord * gamma;
      
      rtReportIntersection(0);
    }
  }
}
