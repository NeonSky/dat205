#include "hip/hip_runtime.h"
#include "common.cuh"

rtBuffer<float4, 2> output_buffer; // RGBA32F

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

RT_PROGRAM void exception() {

  const unsigned int err_code = rtGetExceptionCode();
  if (RT_EXCEPTION_USER <= err_code) {
    rtPrintf("User exception %d at (%d, %d)\n", err_code - RT_EXCEPTION_USER, launch_index.x, launch_index.y);
  } else {
    rtPrintf("Exception code 0x%X at (%d, %d)\n", err_code, launch_index.x, launch_index.y);
  }

  // RGBA32F super magenta as error color (makes sure this isn't accumulated away in a progressive renderer).
  output_buffer[launch_index] = make_float4(1000000.0f, 0.0f, 1000000.0f, 1.0f);
}